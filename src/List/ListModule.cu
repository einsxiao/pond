#include "hip/hip_runtime.h"
#include"pond.h"
#include"ListModule.h"
#include"algorithm"
using namespace pond;
using namespace std;

static INIT_SYMID_OF(Exists);
static INIT_SYMID_OF(exists);
static INIT_SYMID_OF(contain);
static INIT_SYMID_OF(Insert);
static INIT_SYMID_OF(insert);
static INIT_SYMID_OF(Delete);
static INIT_SYMID_OF(delete);
static INIT_SYMID_OF(Append);
static INIT_SYMID_OF(append);
static INIT_SYMID_OF(PushBack);
static INIT_SYMID_OF(push_back);
static INIT_SYMID_OF(push);
static INIT_SYMID_OF(swap);
// static INIT_SYMID_OF(交换);
static INIT_SYMID_OF(Clean);
static INIT_SYMID_OF(clean);
static INIT_SYMID_OF(Length);
static INIT_SYMID_OF(length);
static INIT_SYMID_OF(size );
static INIT_SYMID_OF(reverse );
static INIT_SYMID_OF(sort );
// static INIT_SYMID_OF(添加);
// static INIT_SYMID_OF(插入);
static INIT_SYMID_OF(get);
// static INIT_SYMID_OF(获取);
// static INIT_SYMID_OF(删除);
// static INIT_SYMID_OF(排序);
// static INIT_SYMID_OF(清空);
// static INIT_SYMID_OF(长度);
// static INIT_SYMID_OF(大小);
// static INIT_SYMID_OF(包含);
// static INIT_SYMID_OF(翻转);
// static INIT_SYMID_OF(有);
static INIT_SYMID_OF(has);
static INIT_SYMID_OF(Size);
static INIT_SYMID_OF(SetPart);
static INIT_SYMID_OF(front);
static INIT_SYMID_OF(pop_front);
static INIT_SYMID_OF(back);
static INIT_SYMID_OF(pop_back);
// static INIT_SYMID_OF(头元素);
// static INIT_SYMID_OF(去头);
// static INIT_SYMID_OF(尾元素);
// static INIT_SYMID_OF(去尾);
static INIT_SYMID_OF(items);
// static INIT_SYMID_OF(列表);

__DEBUG_MAIN__("input.pd")

ListModule::ListModule():Module(MODULE_NAME){
  ///////////////////////////////////////////////////////////////////
  {
    RegisterFunction("List",List,this);
    // AddAttribute("List",AttributeType::DelayFunction);
    // AddAttribute("List",AttributeType::DelaySetable);
    AddAttribute("List",AttributeType::Conjunctable);
  }
  {
    RegisterFunction("Dict",Dict,this);
    // AddAttribute("Dict",AttributeType::DelayFunction );
    // AddAttribute("Dict",AttributeType::DelaySetable);
    AddAttribute("Dict",AttributeType::Conjunctable  );
    AddAttribute("Dict",AttributeType::Setable       );
    AddAttribute("Dict",AttributeType::Partable      );
  }
  {
    RegisterFunction("Range",Range,this);
    RegisterFunction("range",Range,this);
    RegisterFunction("序列",Range,this);
  }
  ///////////////////////////////////////////////////////////////////
  {
    RegisterFunction("Take",Take,this); //Take[expr,{i,10}]
    // AddAttribute("Take",AttributeType::GrammarUnit);
  }
  {
    RegisterFunction("Drop",Drop,this); 
    // AddAttribute("Drop",AttributeType::GrammarUnit);
  }
  {
    RegisterFunction("Part",Part,this); 
    AddAttribute("Part",AttributeType::HoldFirst);
    RegisterFunction("SetPart",SetPart,this);
    AddAttribute("SetPart",AttributeType::HoldFirst);
    AddAttribute("Part",AttributeType::Setable );
  }
  ///////////////////////////////////////////////////////////////////
  {
    RegisterFunction("Table",Table,this);
    AddAttribute("Table",AttributeType::HoldAll);
  }
  // AddAttribute("Cases",AttributeType::HoldAll);
  ///////////////////////////////////////////////////////////////////
  RegisterFunction("Head",Head,this); 
  RegisterFunction("表头", Head,this); 
  RegisterFunction("First",First,this); 
  RegisterFunction("头元素",First,this); 
  RegisterFunction("Last",Last,this); 
  RegisterFunction("尾元素",Last,this); 
  RegisterFunction("Most",Most,this); 
  RegisterFunction("去尾",Most,this); 
  RegisterFunction("Rest",Rest,this); 
  RegisterFunction("去头",Rest,this); 
  RegisterFunction("len", (MemberFunction)(&ListModule::PD_Length),this); 
  RegisterFunction("长度", (MemberFunction)(&ListModule::PD_Length),this); 
  RegisterFunction("大小", (MemberFunction)(&ListModule::PD_Length),this); 
  RegisterFunction("bool", (MemberFunction)(&ListModule::PD_Boolean),this); 
  RegisterFunction("布尔值", (MemberFunction)(&ListModule::PD_Boolean),this); 
  ///////////////////////////////////////////////////////////////////
  RegisterFunction("Sort",Sort,this); 
  RegisterFunction("sort",Sort,this); 
  RegisterFunction("排序",Sort,this); 
  RegisterFunction("Flatten",Flatten,this); 
  RegisterFunction("Join",Join,this); 
  RegisterFunction("join",Join,this); 
  RegisterFunction("和并",Join,this); 
  RegisterFunction("Dimensions",Dimensions,this); 
  ///////////////////////////////////////////////////////////////////
  {
    RegisterFunction("Append",Append,this);
    RegisterFunction("AppendTo",AppendTo,this);
    AddAttribute("AppendTo",AttributeType::HoldFirst);
  }
  {
    RegisterFunction("Insert",Insert,this);
    RegisterFunction("InsertTo",InsertTo,this);
    AddAttribute("InsertTo",AttributeType::HoldFirst);
  }
  {
    RegisterFunction("Delete",Delete,this);
    RegisterFunction("DeleteFrom",DeleteFrom,this);
    AddAttribute("DeleteFrom",AttributeType::HoldFirst);
  }
  {
    RegisterFunction("PopBack",PopBack,this);
    AddAttribute("PopBack",AttributeType::HoldFirst);
  }
  {
    RegisterFunction("Reverse",Reverse,this);
    RegisterFunction("翻转",Reverse,this);
    RegisterFunction("Transpose",Transpose,this);
    RegisterFunction("转置",Transpose,this);
  }
  {
    RegisterFunction("NullQ",NullQ,this);
    RegisterFunction("NumberQ",NumberQ,this);
    RegisterFunction("StringQ",StringQ,this);
    RegisterFunction("SymbolQ",SymbolQ,this);
    RegisterFunction("ListQ",ListQ,this);
    // RegisterFunction("曰无",zh_NullQ,this);
    RegisterFunction("曰数字",NumberQ,this);
    RegisterFunction("曰字串",StringQ,this);
    RegisterFunction("曰符号",SymbolQ,this);
    RegisterFunction("曰列表",ListQ,this);
  }
  {
    RegisterFunction("Map",Map,this);
    AddAttribute("Map",AttributeType::HoldFirst);
    RegisterFunction("Apply",Apply,this);
    AddAttribute("Apply",AttributeType::HoldFirst);
    RegisterFunction("Scan",Scan,this);
    AddAttribute("Scan",AttributeType::HoldFirst);
    RegisterFunction("Nest",Nest,this);
    AddAttribute("Nest",AttributeType::HoldFirst);
  }

}
ListModule::~ListModule(){
}

inline bool toNormalIndex(int index,u_int len,int&normalindex){
  if ( index > (int)len or index < -(int)len )
    return false;
  if ( index < 0 )
    normalindex = len +index+1;
  return true;
}

//#define d_return(x) ({dprintf("");return x; })
bool SpanToIndexes(int len,Object &index, int &start, int &end, int &step ){
  //dout<<"try deal with span = "<< index <<endl;
  if ( not (index.Size() == 3 or index.Size() == 2 ) )
    return(false);
  //step 
  bool givenStep = false;
  if ( index.Size() == 2 ){
    step = 1;
  }else if ( index[3].NumberQ( ) ){
    step = (int)index[3];
    givenStep = true;
    if ( step == 0 )
      return(false);
  }else if ( index[3].NullQ() ){
    step = 1;
  }else
    return(false);
  //start
  if ( index[1].NullQ() ){
    if ( step > 0 )
      start = 1;
    else
      start = len;
  }else if ( index[1].NumberQ( ) ){
    start = (int)index[1];
    if ( start > len or start < - len )
      return(false);
    if ( start < 0 ) start += len +1;
  }else
    return(false);
  //end
  if ( index[2].NullQ() ){
    if ( step > 0 )
      end = len;
    else
      end = 1;
  }else if ( index[2].NumberQ( ) ){
    end = (int)index[2];
    if ( end > len or end < -len ) return(false);
    if ( end < 0 ) end += len +1;
  }else return(false);

  if ( givenStep ) return true;
  if ( start > end and step > 0 ){
    step = -step;
  }
  return true;
}
//#undef return

void localSetPart(Object&valueList,Object&partList,u_int pos,Object&rightValue);

int ListModule::List(Object &ARGV){
  //cout<<"deal List:"<< ARGV <<endl;
  // DelaySet_Context(List){ // 
  //   //dout<<"list delay set context"<<endl;
  //   // List[...][...] = 
  //   // |  1,0 |  1,1 |   2
  //   Object res;
  //   localSetPart(ARGV[1][0], ARGV[1], 1, ARGV[2] );
  //   ARGV = ARGV[2];
  //   ReturnNormal;
  // }
  // DelayFunction_Context(List){
  //   //dout<<"list delay function context"<<endl;
  //   Object res;
  //   GetPartList( ARGV[0], ARGV, 1, res, true );
  //   ARGV = res;
  //   ReturnNormal;
  // }
  Conjunct_Context(List){
    //dout<<"list conjunct context"<<endl;
    Object &list = ARGV[1];
    Object &oper = ARGV[2];

    Conjunct_Case2( push, append ){
      CheckArgsShouldEqual( oper, 1 );
      EvaKernel->Evaluate( oper[1] );
      list.PushBackCopy( oper[1] );
      ReturnObject(list);
    }
    Conjunct_Case( insert ){
      CheckArgsShouldEqual( oper, 2 );
      EvaKernel->Evaluate( oper[1] );
      EvaKernel->Evaluate( oper[2] );

      CheckArgsShouldBeNumber( oper, 1 );
      list.InsertCopy( int(oper[1]), oper[2] );
      ReturnObject(list);
    }
    Conjunct_Case( swap ){
      CheckArgsShouldEqual( oper, 2 );
      EvaKernel->Evaluate( oper[1] );
      EvaKernel->Evaluate( oper[2] );
      CheckArgsShouldBeNumber( oper,1);
      CheckArgsShouldBeNumber( oper,2);
      int p1 = oper[1].Number();
      int p2 = oper[2].Number();
      if ( p1 != p2 ) 
        list[p1].Swap( list[p2] );
      ReturnObject(list);
    }
    Conjunct_Case2(has, exists ){
      CheckArgsShouldEqual(oper,1);
      for (auto iter = list.Begin(); iter != list.End(); iter ++ ){ 
        //dout<<" do test "<< (*iter) << "  ?=  "<< oper<<endl;
        if ( (*iter) == oper[1]  ){
          ReturnBoolean(true);
        }
      }
      ReturnBoolean(false);
    }
    Conjunct_Var_Case( size ){
      ReturnNumber( list.Size() ); 
    }
    Conjunct_Case2( length, size ){
      CheckArgsShouldEqual( oper, 0 );
      ReturnNumber( list.Size() ); 
    }
    Conjunct_Case( delete ){
      CheckArgsShouldEqual( oper, 1 );
      EvaKernel->Evaluate( oper[1] );
      CheckArgsShouldBeNumber( oper, 1 );
      int ind = int( oper[1] );
      if ( ind < 1 or ind > (int)list.Size() ) {
        zhErroring("列表::删除","下标 "+ToString(ind)+" 超出有效范围.") ||
          _Erroring("List::Delete","Index "+ToString(ind)+" is out of range.");
        return -1;
      }
      list.Delete(ind);
      ReturnObject(list);
    }
    Conjunct_Case( clean ){
      CheckArgsShouldEqual( oper, 0 );
      list.DeleteElements();
      ReturnObject(list);
    }
    Conjunct_Case( reverse ){
      CheckArgsShouldEqual( oper, 0 );
      list.Reverse();
      ReturnObject(list);
    }
    Conjunct_Case( sort ){
      CheckArgsShouldEqual( oper, 0 );
      list.Sort();
      ReturnObject(list);
    }
    Conjunct_Case( front ){
      CheckArgsShouldEqual( oper, 0 );
      ReturnObject(list[1]);
    }
    Conjunct_Case( pop_front ){
      CheckArgsShouldEqual( oper, 0 );
      list.Delete(1);
      ReturnObject(list);
    }
    Conjunct_Case( back ){
      CheckArgsShouldEqual( oper, 0 );
      ReturnObject( list.Back() );
    }
    Conjunct_Case( pop_back ){
      CheckArgsShouldEqual( oper, 0 );
      list.PopBack();
      ReturnObject(list);
    }
    //If no case matched report error 
    Object l = ( oper.ListQ() ? oper[0] : oper );
    zhErroring("列表",l.ToString() + " 不是列表方法" )||
      _Erroring("List",l.ToString() + " is not a List method.");
    ReturnError;
  }

  //dealing Sequence
  u_int i = 1;
  while ( i<= ARGV.Size() ){
    if ( ARGV.ElementAt(i).ListQ( pond::SYMID_OF_Sequence  ) ){
      ARGV.InsertRef( ARGV.Begin()+i, ARGV.ElementAt(i).Begin(), ARGV.ElementAt(i).End() );
      ARGV.Delete(i);
    }else{
      EvaKernel->Evaluate( ARGV[i], false, true );
      i++;
    }
  }
  return 1;
}

int ListModule::range_iter(Object&ARGV,long&N,double&start,double&incr){
  CheckShouldBeWithin(1,3);
  if ( ARGV.Size() == 1 ){
    EvaKernel->Evaluate( ARGV[1] );
    if ( ARGV[1].NumberQ() ){
      start  = 1;
      N      = ARGV[1].Number();
      incr   = 1;
      return 1;
    }
    return 0;
  }

  if ( ARGV.Size() == 2 ){
    EvaKernel->Evaluate(ARGV[1]);
    EvaKernel->Evaluate(ARGV[2]);
    if ( ARGV[1].NumberQ() && ARGV[2].NumberQ() ){
      start = ARGV[1].Number();
      N     = ARGV[2].Number() - start+1;
      incr  = 1;
      return 1;
    }
    return 0;
  }

  if ( ARGV.Size() == 3 ){
    EvaKernel->Evaluate(ARGV[1]);
    EvaKernel->Evaluate(ARGV[2]);
    EvaKernel->Evaluate(ARGV[3]);
    if ( ARGV[1].NumberQ() && ARGV[2].NumberQ() && ARGV[3].NumberQ() ){
      start =ARGV[1].Number() ;
      incr  =ARGV[3].Number() ;
      if ( incr == 0 ){
        zhErroring("range","range的增量不能为零.")||
          _Erroring("range","Increment should not be a zero.");
        return 0;
      }
      N     =(ARGV[2].Number() - start)/incr + 1;
      return 1;
    }
    return 0;
  }
  return 0;
}

int ListModule::Range(Object &ARGV){
  double start, incr; long N;
  if ( ListModule::range_iter(ARGV,N,start,incr) == 0 ){
    zhErroring("for:in:range","range 不合法") ||
      _Erroring("for:in:range","range is not valid.");
    ReturnError;
  }
  ARGV.SetList();
  ARGV.ReserveSize( N );
  for ( int i = 0; i< N; i++ ){
    ARGV.PushBackNumber(start+i*incr);
  }
  ReturnNormal;
}

static int take(Object&expr,Object&takelist,u_int p,Object&resultList){//expr should make sure to be a nonatom
  Object & iter = takelist[p];
  resultList.SetList();
  if ( iter.NumberQ() ){
    int N =(double) iter;
    if ( N >= 0 ){
      if ( N > (int)expr.Size() ){
        { _Erroring("Take","Cannot take position 1 through "+ToString(N)+"."); ReturnError; }
      }
      for ( int i = 1; i<= N; i++ ){
#define address_ele_take(p_i)                                   \
        if ( p == takelist.Size() ){                            \
          resultList.PushBackRef( expr[p_i] );                  \
        }else{                                                  \
          resultList.PushBackNull();                            \
          take( expr[p_i], takelist, p+1, resultList.Back() );  \
        }                                                       \
        /////
        address_ele_take(i);
      }
    }else{
      if ( -N> (int)expr.Size() ){
        { _Erroring("Take","Cannot take position "+ToString(N)+"through -1."); ReturnError; }
      }
      for ( int i = expr.Size() + N+1; i<= (int)expr.Size(); i++ ){
        address_ele_take(i);
      }
    }
    return 1;
  }
  if ( (iter).ListQ(SYMID_OF_List) ){//AtomQ must be not true
    if ( iter.Size() > 3 || iter.Size()< 1 ){
      { _Erroring("Take","Sequence specification "+iter.ToString()+" is not valid."); ReturnError; }
    }
    Object &num1 = (iter)[1];
    int ind1,bak;
    if ( num1.NumberQ() ){
      bak = ind1 =(double) num1;
      if ( ind1<0 ) ind1 += expr.Size() +1 ; 
      if ( iter.Size() == 1){//only one number...
        if ( ind1 > (int)expr.Size() or ind1 < 1 ){
          { _Erroring("Take","Cannot take position "+ToString(bak) ); ReturnError; }
        }
        address_ele_take( ind1 );
        return 1;
      }
    }else{
      { _Erroring("Take","Sequence specification "+iter.ToString()+" is not valid."); ReturnError; }
    }
    Object&num2 = (iter)(2);
    int ind2;
    if ( num2.NumberQ() ){
      bak = ind2 =(double) num2;
      if ( ind2<0 ) ind2 += expr.Size()+1;
      if ( iter.Size() == 2){
        if ( ind2 > (int)expr.Size()  or ind2 < 1 ){
          { _Erroring("Take","Cannot take position "+ToString(bak)+" through "+ToString(bak)+"."); ReturnError; }
        }
        for ( int i=ind1; i<= ind2; i++ ){
          address_ele_take( i );
        }
        return 1;
      }
    }else{
      { _Erroring("Take","Sequence specification "+iter.ToString()+" is not valid."); ReturnError; }
    }
    Object &inter = (iter)(3);
    int ninter;
    if ( inter.NumberQ() ){
      ninter =(double)inter;
      if ( iter.Size() == 3){
        if ( ninter == 0 ){
          { _Erroring("Take","Sequence specification "+iter.ToString()+" is not valid."); ReturnError; }
        }
        if ( ninter> 0 ){
          for ( int i = ind1; i <= ind2; i+=ninter ){
            address_ele_take(i);
          }
        }else if ( ninter < 0 ){
          for ( int i = ind1; i>= ind2; i+=ninter ){
            address_ele_take(i);
          }
        }
        return 1;
      }
    }else{
      { _Erroring("Take","Sequence specification "+iter.ToString()+" is not valid."); ReturnError; }
    }
  }
  { _Erroring("Take","Sequence specification "+iter.ToString()+" is not valid."); ReturnError; }
}

int ListModule::Take(Object &ARGV){
  CheckArgsShouldNoLessThan(ARGV,2);
  CheckShouldBeList(1);
  
  Object resultList;
  take( ARGV[1], ARGV, 2, resultList ); 
  ARGV = resultList;
  ReturnNormal;
}

static int drop(Object&expr, Object&droplist,u_int p){
  Object &iter = droplist[p];
  if ( iter.AtomQ() ){
    if ( iter.NumberQ() ){
      int N = iter.Number();
      if ( N>=0 ){
        if ( N> (int)expr.Size() ){ _Erroring("Drop","Cannot drop position 1 through "+ToString(N)+" in "+expr.ToString()+"."); ReturnError; }
        expr.Delete( expr.Begin(),expr.Begin()+N );
#define address_next_level_of_drop_if_exist                             \
        if ( p < droplist.Size() ){                                     \
          for (auto iter = expr.Begin(); iter != expr.End(); iter ++ ){ \
            drop ( *iter, droplist, p+ 1 );                             \
          }                                                             \
        }
        address_next_level_of_drop_if_exist;
      }else{
        if ( -N> (int)expr.Size() ){{ _Erroring("Drop","Cannot drop position "+ToString(N)+" through -1 in "+expr.ToString()+".");} ReturnError; }
        expr.Delete( expr.End() + N, expr.End() );
        address_next_level_of_drop_if_exist;
      }
      return 1;
    }else{
      { _Erroring("Drop","Sequence specification "+iter.ToString()+" is not valid."); ReturnError; }
    }
  }
  if ( (iter).ListQ(SYMID_OF_List) ){//AtomQ must be not true
    if ( iter.Size() > 3 || iter.Size()< 1 ){{ _Erroring("Drop","Sequence specification "+iter.ToString()+" is not valid.");} ReturnError; }
    Object &num1 = (iter)(1);
    int ind1,indb1;
    if ( num1.NumberQ() ){
      indb1 = ind1 =  num1.Number() ;
      if ( ind1<0 ) ind1 += expr.Size()+1;
      if ( iter.Size() == 1){
        if ( abs(ind1) > (int)expr.Size() ){{ _Erroring("Drop","Cannot drop position "+ToString(indb1)+" in "+expr.ToString()+"."); } ReturnError; }
        expr.Delete(ind1);
        address_next_level_of_drop_if_exist;
        return 1;
      }
    }else{
      { _Erroring("Drop","Sequence specification "+iter.ToString()+" is not valid."); ReturnError; }
    }
    Object&num2 = (iter)(2);
    int ind2,indb2;
    if ( num2.NumberQ()  ){
      indb2 = ind2 =  num2.Number();
      if ( ind2<0 ) ind2 += expr.Size()+1;
      if ( iter.Size() == 2){
        if ( abs(ind2) > (int)expr.Size() ){{ _Erroring("Drop","Cannot drop position "+ToString(indb1)+" through "+ToString(indb2)+" in "+expr.ToString()+".");} ReturnError; }
        for (int i = ind1; i<= ind2; i++ )
          expr.Delete(ind1);
        address_next_level_of_drop_if_exist;
        return 1;
      }
    }else{
      { _Erroring("Drop","Sequence specification "+iter.ToString()+" is not valid."); ReturnError; }
    }
    Object &inter = (iter)(3);
    int ninter;
    if ( inter.IntegerQ() ){
      ninter =  inter.Number();
      if ( iter.Size() == 3){
        if ( (int)inter == 0 ){ _Erroring("Drop","Sequence specification "+iter.ToString()+" is not valid."); ReturnError; }
        if ( ninter> 0 ){
          int adj = ( ind2 - ind1 ) % ninter ;
          for ( int i = ind2-adj; i >= ind1; i-=ninter ){
            expr.Delete( i );
          }
          address_next_level_of_drop_if_exist;
        }else if ( ninter < 0 ){
          for ( int i = ind1; i>= ind2; i+=ninter ){
            expr.Delete( i );
          }
          address_next_level_of_drop_if_exist;
        }
        return 1;
      }
    }
    _Erroring("Drop","Sequence specification "+iter.ToString()+" is not valid.");
    ReturnError; 
  }
  _Erroring("Drop","Sequence specification "+iter.ToString()+" is not valid.");
  ReturnError;
}

int ListModule::Drop(Object &ARGV){
  CheckArgsShouldNoLessThan(ARGV,2);
  CheckShouldBeList(1);
  Object expr = ARGV(1);
  if ( expr.AtomQ() ){
    _Erroring("Drop","Nonatomic experssion expected at position 1.");
    ReturnError;
  }
  drop( ARGV[1], ARGV, 2 );
  ARGV = ARGV[1];
  ReturnNormal;
}

// for obj = Part[ {...}, p1, p2]
// valueList is obj[1], partList is obj,  pos start from 2 (last evaluated position)
int ListModule::GetPartList(Object&valueList,Object&partList,u_int pos,Object&resultList,bool isReal){
  if ( valueList.NullQ() or partList.NullQ() ) return -1;
  if ( valueList.AtomQ() ) {
    zhErroring("索引运算","不能对原子量 "+valueList.ToString()+" 进行索引运算.")||
      _Erroring("Part","Part specification for atom "+valueList.ToString()+" is not allowed.");
    ReturnError;
  }
  bool LastLevel = ( partList.Size() == pos );
  Object &index = partList[pos]; 
  if ( index.ListQ() ){
    if ( index.ListQ(SYMID_OF_List) ){
      resultList.SetList();
      for ( u_int i =1; i<= index.Size(); i++ ){
        if ( not (index)[i].NumberQ() ){
          zhErroring("索引运算","索引下标应该是数字")||
            _Erroring("Part","Part specification should be a number");
          ReturnError;
        }
        int ind = index[i].Number();
        if ( not toNormalIndex(ind,valueList.Size(),ind) ){
          zhWarning("索引运算",valueList.ToString()+" 索引下标 "+ToString(ind)+" 超出范围.") ||
            _Warning("Part","Part specification "+ToString(ind)+" for "+valueList.ToString()+" is out of range.");
          return -1;
        }
        if ( LastLevel ){
          if ( isReal ){
            //dprintf(" do a real pushback ");
            resultList.PushBackRef( valueList[ind] ); 
          } else {
            //dprintf(" do a copy pushback ");
            resultList.PushBackCopy( valueList[ind] ); 
          }
        }else{
          resultList.PushBackNull();
          GetPartList(valueList(ind),partList,pos+1,resultList[i],isReal);
        }
      }
      return 1;
    }else if ( index.ListQ( SYMID_OF_Span ) ){
      int start, end, step;
      resultList.SetList();
      if ( not  SpanToIndexes( valueList.Size(), index, start, end, step ) ){ // just return {}
        ReturnNormal;
        // _Erroring("Span","Span "+index.ToString()+" is in the wrong form.");
        // ReturnError;
      }
      //dprintf("%s -> start = %d, end = %d, step = %d",index.ToString().c_str(),start,end,step);
      for ( int i = start; (step>0?i<=end:i>=end); i+=step ){
        if (LastLevel ){
          if ( isReal ){
            //dprintf(" do a real pushback ");
            resultList.PushBackRef( valueList[ i ] );
          }else{
            //dprintf(" do a copy pushback ");
            resultList.PushBackCopy( valueList[ i ] );
          }
        }else{
          resultList.PushBackNull();
          GetPartList( valueList[ i ], partList, pos+1, resultList[i], isReal);
        }
      }
      return 1;
    }
    zhErroring("索引运算"," 索引下标格式错误.") ||
      _Erroring("Part","Part specification is in wrong form.");
    ReturnError; 
  }
  if ( not partList[pos].NumberQ() )
    return -1;
  int ind = partList[pos].Number();
  if ( not toNormalIndex(ind,valueList.Size(),ind) ){

    zhWarning("索引运算",valueList.ToString()+" 索引下标 "+ToString(ind)+" 超出范围.") ||
      _Warning("Part","Part specification "+ToString(ind)+" for "+valueList.ToString()+" is out of range.");
    return -1;
  }
  if ( LastLevel ){
    if ( isReal ){
      //dprintf(" do a real replace");
      resultList =  valueList[ind] ;
    }else{
      //dprintf(" do a copy replace");
      resultList.CopyObject( valueList[ind] );
    }
  }else{
    GetPartList( valueList[ind], partList, pos+1, resultList, isReal);
  }
  return 1;
}

int ListModule::Part(Object&ARGV){
  //cout<<"into Part with ARGV ="<<ARGV<<endl;
  Set_Context(Part){
    //dout<<"into Part set context with ARGV ="<<ARGV<<endl;
    return SetPart( ARGV );
  }
  CheckShouldNoLessThan(2);
  // rewrite the Part[Part[a_,b__],c__] :> Part[a,b,c]
  while ( ARGV[1].ListQ( SYMID_OF_Part ) ){ //flaten the part
    CheckArgsShouldNoLessThan(ARGV[1],2);
    ARGV.InsertRef(ARGV.Begin()+1,ARGV[1].Begin(),ARGV[1].End());
    ARGV.Delete(1);
  }
  // the value of part should be evaluated till last
  //cerr<<"ori valueList ="<< ARGV[1].ToFullFormString() <<endl;
  EvaKernel->Evaluation::Evaluate( ARGV[1], false, true ); 
  // check if Partable, and deternmine whether send to Partable one
  //dout<<"check if is partable argv="<< ARGV<<endl;
  if ( ARGV[1].ListQ() ){
    EvaRecord * rec = EvaKernel->GetEvaRecord( ARGV[1][0] );
    if ( rec and AttributeQ( rec->attributes , AttributeType::Partable) ){
      // the Part pass through has form
      // Part( f(...), index... )           f is referenced value if is from value pair
      //dout<< "pass "<<ARGV<<" to "<< ARGV[1][0] <<endl;
      return EvaKernel->Call( rec, ARGV );
    }
  }
  Object resultList;
  // here, part list should not be real
  //if ( GetPartList(ARGV[1],ARGV,2,resultList,false) < 0 ) ReturnHold; 
  if ( GetPartList(ARGV[1], ARGV, 2, resultList, true) < 0 ) ReturnHold; 
  ARGV = resultList;
  //dout<<" after part is:"<<ARGV<<endl;
  // extra evaluate need to be done
  // e.g.  funcs = {x^2,y^2}   funcs[[1]]
  EvaKernel->Evaluate(ARGV); 
  //dout<<" after evaluate:"<<ARGV<<endl;
  ReturnNormal;
}

void localSetPart(Object&valueList,Object&partList,u_int pos,Object&rightValue){
  if ( valueList.NullQ() or partList.NullQ() ) return;
  if ( valueList.AtomQ() ) {
    zhErroring("索引运算","对原子类型的值 "+valueList.ToString()+" 进行索引运算不太合适.") ||
      _Erroring("Part","Part specification for atom "+valueList.ToString()+" is not allowed.");
    return;
  }
  bool LastLevel = ( partList.Size() == pos );
  Object&index = partList[pos];
  EvaKernel->Evaluate( index );
  if ( index.ListQ() ){
    if ( index.ListQ(SYMID_OF_List) ){
      if ( rightValue.ListQ(SYMID_OF_List) ){
        if ( index.Size() != rightValue.Size() ) {
          zhErroring("索引赋值","赋值左右两边列表维度不同.")||
            _Erroring("SetPart","Left Part List should have the size of the right value List.");
          return;
        }
        for ( u_int i=1; i<= index.Size(); i++ ){
          if ( not (index)[i].NumberQ() ){
            zhErroring("索引运算","所引运算必须使用数字")||
              _Erroring("Part","Part specification should be a number");
            return;
          }
          int ind = (index)[i].Number();
          if ( not toNormalIndex(ind,valueList.Size(),ind) ){
            _Erroring("索引运算","索引号 "+ToString(ind)+" 所指向的元素不存在") ||
              _Erroring("Part","Part "+ToString(ind)+" of list does not exist.");
            return;
          }
          if ( LastLevel ){
            valueList[ind].CopyObject( rightValue[i] );
          }else{
            localSetPart( valueList[ind], partList,pos+1,rightValue[i] );
          }
        }
        return;
      }
      //only an normal Object
      for ( u_int i =1; i<= index.Size(); i++ ){
        if ( not (index)[i].NumberQ() ){
          zhErroring("索引运算","所引运算必须使用数字")||
            _Erroring("Part","Part specification should be a number");
          return;
        }
        int ind = (index)[i].Number();
        if ( not toNormalIndex(ind,valueList.Size(),ind) ){
          _Erroring("索引运算","索引号 "+ToString(ind)+" 所指向的元素不存在") ||
            _Erroring("Part","Part "+ToString(ind)+" of list does not exist.");
          return;
        }
        if ( LastLevel ){
          valueList[ind].CopyObject( rightValue );
        }else{
          localSetPart( valueList[ind],partList,pos+1,rightValue);
        }
      }
      return;
    }else if ( index.ListQ( SYMID_OF_Span ) ){
      int start, end, step;
      if ( not  SpanToIndexes( valueList.Size(), index, start, end, step ) ) {
        zhErroring( "范围索引","范围索引 "+index.ToString()+" 格式错误.");
        _Erroring("Span","Span "+index.ToString()+"in not in the right form.");
        return;
      }
      for( int i = start; (step>0?i<= end:i>=end); i+=step ){
        if ( LastLevel )
          valueList[i].CopyObject( rightValue );
        else
          localSetPart( valueList[i], partList, pos+1, rightValue );
      }
      return;
    }
    zhErroring( "索引赋值","索引值 "+index.ToString()+" 格式错误.") ||
    _Erroring("SetPart","Part index "+index.ToString()+" is not in the right form.");
    return;
  }
  if ( not index.NumberQ() ) {
    zhErroring("索引运算","索引运算必须使用数字")||
      _Erroring("Part","Part specification should be a number");
    return;
  }
  int ind = index.Number();
  if ( not toNormalIndex(ind,valueList.Size(),ind) ){
    zhErroring("索引运算","索引值 "+ToString(ind)+" 所指元素不存在.")||
      _Erroring("Part","Part "+ToString(ind)+" of "+valueList.ToString()+" does not exist.");
    return;
  }
  if ( LastLevel ){
    valueList[ind].CopyObject( rightValue );
  }else{
    localSetPart( valueList[ind],partList,pos+1,rightValue);
  }
  return;
}

int ListModule::SetPart(Object&part,Object &rightValue){
  if ( not part.ListQ( SYMID_OF_Part ) ) {
    zhErroring("索引赋值","错误调用索引赋值函数.") ||
      _Erroring("SetPart","Invalid calling for SetPart.");
    ReturnError;
  }
  CheckArgsShouldNoLessThan(part, 2);
  while ( part[1].ListQ( SYMID_OF_Part ) ){
    CheckArgsShouldNoLessThan( part[1],2 );
    part.InsertRef( part.Begin()+1, part[1].Begin(),part[1].End() );
    part.Delete(1);
  }
  Object value = EvaKernel->GetValuePairRef( part[1] );
  //cout<<"in set part part value =" << part[1] << ":"<< value <<endl;
  if ( value.NullQ() ) {
    // check whether part[1] itself is setable
    EvaRecord *rec = EvaKernel->GetEvaRecord( part[1][0] );
    //cout<<" rec for part[1][0] "<< part[1][0] << ":"<<rec <<endl;
    if ( rec and AttributeQ( rec->attributes , AttributeType::Setable ) ){
      //cout<<" rec for part is setable"<<endl;
      Object ARGV(ObjectType::List, SYMID_OF_Set );
      ARGV.PushBackRef( part ).PushBackRef( rightValue ).PushBackRef( part[1] );
      return EvaKernel->Call(rec, ARGV );
    } else {
      zhErroring("索引赋值",part[1].ToString()+" 不是一个被赋值的符号.") ||
        _Erroring("SetPart",part[1].ToString()+" is not an Symbol with a value.");
      ReturnError;
    }
  }else if ( not value[2].ListQ() ){
    zhErroring("索引赋值",part[1].ToString()+" 的值不是一个列表.") ||
      _Erroring("SetPart","value of "+part[1].ToString()+" is not a list.");
    ReturnError;
  }
  // check whether setable 
  EvaKernel->Evaluate( value[2][0] );
  if ( value[2][0].SymbolQ() ){ // a vlaued symbol
    EvaRecord * rec = EvaKernel->GetEvaRecord( value[2][0] );;
    if ( rec and AttributeQ( rec->attributes , AttributeType::Setable ) ){
      //cout<<"rec is setable "<< value[2] << value[2][0]  <<endl;
      Object ARGV(ObjectType::List,SYMID_OF_Set );
      ARGV.PushBackRef( part )     // part 
        .PushBackRef( rightValue ) // right value
        .PushBackRef( value[2] );  // dict ref
      //dout<<" pass "<<  ARGV  <<" to "<<  value[2][0] <<endl;
      return EvaKernel->Call(rec, ARGV );
    }
  }
  localSetPart( value[2], part, 2, rightValue );
  return 0;
}

int ListModule::SetPart( Object&ARGV){
  CheckShouldEqual( 2 );
  //cout<<"into set part = "<< ARGV.ToFullFormString() << endl;
  SetPart( ARGV[1], ARGV[2]);
  ARGV = ARGV[2];
  ReturnNormal;
}

///////////////////////////////////
int ListModule::Head(Object &ARGV){
  CheckShouldEqual(1);
  if ( ARGV[1].ListQ() ){
    Object head = ARGV[1][0];
    ARGV = head;
    ReturnNormal;
  }else{
    const char * head = ARGV[1].Head();
    ARGV.SetSymbol( head );
    ReturnNormal;
  }
}

int ListModule::First(Object &ARGV){
  CheckShouldEqual(1);
  CheckArgsShouldBeList(ARGV,1);
  if ( ARGV[1].Size()<1 ){
    _Warning("First","Try to get first element of an empty List");
    ReturnNull;
  }
  ARGV = ARGV[1][1];
  ReturnNormal;
}

int ListModule::Last(Object &ARGV){
  CheckShouldEqual(1);
  CheckShouldBeList(1 );
  if ( ARGV[1].Size()<1 ){
    _Warning("Last","Try to get last element of an emtpy List");
    ReturnNull;
  }
  ARGV = ARGV[1][ARGV[1].Size() ];
  ReturnNormal;
}
int ListModule::Most(Object &ARGV){
  CheckShouldEqual(1);
  CheckShouldBeList(1);
  if ( ARGV[1].Size()>1 ){
    ARGV[1].PopBack();
  }
  ARGV = ARGV[1];
  ReturnNormal;
}

int ListModule::Rest(Object &ARGV){
  CheckShouldEqual(1);
  CheckShouldBeList(1 );
  //dout<<"Rest with argv = "<< ARGV <<endl;
  if ( ARGV[1].Size()>1 ){
    ARGV[1].Delete(1);
  }
  ARGV = ARGV[1];
  ReturnNormal;
}
///////////////////////////////////

static INIT_SYMID_OF( Table );
int ListModule::Table(Object &ARGV){
  CheckArgsShouldNoLessThan(ARGV,2);
  //dout<<"come to table with argv = "<<ARGV<<endl;
  while ( ARGV.Size() >2 ){//change Table to recursively form
    Object newtable; newtable.SetList( SYMID_OF_Table );
    newtable.PushBackRef( ARGV(1) );
    newtable.PushBackRef( ARGV(2) );
    ARGV.Delete(2);
    ARGV(1)=newtable;
  }
  Object&expr = ARGV[1];
  Object&iter = ARGV[2];
  if ( not (iter).ListQ() ) EvaKernel->Evaluate( iter );
  if ( not (iter).ListQ() || iter.Size()<1 || iter.Size()>4 ) {
    _Erroring(ARGV.Key(),"Argument "+iter.ToString()+" is not an iterator.\n");
    ReturnError;
  }
  if ( iter.Size() == 1 ){
    Object&num = (iter)(1);
    EvaKernel->Evaluate( num );
    if ( num.NumberQ() ){
      Object newlist; newlist.SetList();
      int N =(double)num;
      newlist.ReserveSize(N);
      for ( int i = 1; i<=N; i++ ){
        Object newexpr; newexpr.CopyObject( expr );
        EvaKernel->Evaluate( newexpr );
        newlist.PushBackRef( newexpr );
      }
      ARGV = newlist;
      ReturnNormal;
    }
    { _Erroring(ARGV.Key(),"Iterator "+iter.ToString()+" does not have appropriate bounds."); ReturnError; }
  }
  if ( iter.Size() == 2 ){
    Object&var = (iter)(1);
    if ( !var.SymbolQ() ) { _Erroring("Do",var.ToString()+" cannot be used as an iterator."); ReturnError; }
    Object&num = (iter)(2);
    EvaKernel->Evaluate( num );
    Object&vobj = EvaKernel->StackPushCopy(var,var)[2];
    if ( num.NumberQ() ){ // form:  {i, 100}
      Object newlist; newlist.SetList();
      int N =(double)num;
      newlist.ReserveSize(N);
      for ( int i = 1; i<=N; i++ ){ 
        //dout<<"try loop "<<i<<endl;
        vobj.SetNumber( i );
        //dout<< "vobj = "<<vobj <<endl;
        //dout<< " stack = "<< EvaKernel->currentValueTable->stackTable << endl;
        //dout<< " test getvalue for i = "<< EvaKernel->GetValuePairRef( tmp ) << endl;
        Object newexpr; newexpr.CopyObject( expr );
        //dout<<" with newexpr ="<<newexpr<<endl;
        EvaKernel->Evaluate( newexpr );
        newlist.PushBackRef( newexpr );
      }
      ARGV = newlist;
      //dout<<"result:"<<ARGV<<endl;
      EvaKernel->StackPop();
      //dout<<"result after pop:"<<ARGV<<endl;
      ReturnNormal;
    }else if ( (num).ListQ()){  // form:   {i, { 1,3,5, 9 , 20} }
      Object&vobj= EvaKernel->StackPushCopy(var,var)[2];
      Object newlist; newlist.SetList();
      for ( u_int i = 1; i<=num.Size(); i++ ){
        Object newexpr; newexpr.CopyObject( expr );
        vobj.SetNumber( i );
        EvaKernel->Evaluate( newexpr );
        newlist.PushBackCopy( newexpr );
      }
      ARGV =  newlist ;
      EvaKernel->StackPop();
      ReturnNormal;
    }
    _Erroring(ARGV.Key(),"Iterator "+iter.ToString()+" does not have appropriate bounds."); 
    ReturnError; 
  }
  if ( iter.Size() == 3 ){ // form:  {i, 1, 100}
    Object&var = (iter)(1);
    if ( !var.SymbolQ() ) { _Erroring(ARGV.Key(),var.ToString()+" cannot be used as an iterator."); ReturnError; }
    Object&num1 = (iter)(2);
    Object&num2 = (iter)(3);
    EvaKernel->Evaluate( num1 );
    EvaKernel->Evaluate( num2 );
    if ( num1.NumberQ() && num2.NumberQ() ){
      Object&vobj = EvaKernel->StackPushCopy(var,var)[2];
      Object newlist; newlist.SetList();
      int Nbegin = num1.Number() ;
      int Nend =  num2.Number();
      for ( int i = Nbegin; i<=Nend; i++ ){
        Object newexpr; newexpr.CopyObject(expr);
        vobj.SetNumber(i);
        EvaKernel->Evaluate( newexpr );
        newlist.PushBackCopy( newexpr );
      }
      ARGV = newlist;
      EvaKernel->StackPop();
      ReturnNormal;
    }
    _Erroring(ARGV.Key(),"Iterator "+iter.ToString()+" does not have appropriate bounds."); 
    ReturnError;
  }
  if ( iter.Size() == 4 ){ // form:  {i, 1, 100, 3}
    Object&var = (iter)(1);
    if ( !var.SymbolQ() ) { _Erroring(ARGV.Key(),var.ToString()+" cannot be used as an iterator."); ReturnError; }
    Object&num1 = (iter)(2);
    Object&num2 = (iter)(3);
    Object&incr = (iter)(4);
    //dout<<"current var iter is "<<var<<endl;
    EvaKernel->Evaluate( num1 );
    EvaKernel->Evaluate( num2 );
    EvaKernel->Evaluate( incr );
    if ( num1.NumberQ() && num2.NumberQ() && incr.NumberQ() ){
      Object pair = EvaKernel->StackPushCopy(var,var);
      //dout<<"value pair is"<<pair<<endl;
      Object newlist; newlist.SetList();
      double Nbegin = num1.Number();
      double Nend = num2.Number();
      double Nincr = incr.Number();
      if ( Nincr == 0 ) { _Erroring("Table","Increment should not be a zero."); ReturnError; }
      int steps = (Nend-Nbegin)/Nincr;
      for ( int i = 0; i<= steps; i++ ){
        Object newexpr; newexpr.CopyObject(expr);
        //dout<<"loop "<<i<<" for expr: "<< newexpr<<endl;
        pair[2].SetNumber(Nbegin+i*Nincr);
        //dout<<"value pair is "<<pair<<endl;
        EvaKernel->flag = 1;
        //dout<<"kernel pointer = "<<EvaKernel<<endl;
        EvaKernel->Evaluate( newexpr );
        EvaKernel->flag = 0;
        //dout<<"value pair after evaluate "<<pair<<endl;
        newlist.PushBackCopy( newexpr );
      }
      ARGV =  newlist ;
      EvaKernel->StackPop();
      ReturnNormal;

    }
    { _Erroring(ARGV.Key(),"Iterator "+iter.ToString()+" does not have appropriate bounds."); ReturnError; }
  }
  ReturnHold;
}

int ListModule::Sort(Object &ARGV){
  CheckArgsShouldEqual(ARGV,1);
  if ( not ARGV[1].ListQ() ) {
    _Erroring(ARGV.Key(),"1st argument of "+ARGV.ToString()+" should be a ARGV.");
    ReturnError;
  }
  ARGV[1].ExpressionSort();
  ARGV = ARGV[1];
  ReturnNormal;
}

int ListModule::Flatten(Object &ARGV){
  CheckArgsShouldEqual(ARGV,1);
  ARGV = ARGV[1];
  ARGV.Flatten();
  ReturnNormal;
}

int ListModule::Join(Object &ARGV){
  //dout<<"try to do Join with ARGV = "<<ARGV<<endl;
  if ( ARGV.Size() < 1  ){
    ARGV.SetList( SYMID_OF_List );
    ReturnNormal;
  }
  bool is_dict = true;
  for (int i=1; i<= ARGV.Size(); i++){
    if ( not ARGV[i].ListQ(SYMID_OF_Dict) and not ARGV[i].NullQ() ){
      is_dict = false;
      break;
    }
  }
  if ( is_dict ){
    Object&ndict = ARGV[1];
    for (int i = 2; i<= ARGV.Size(); i++){
      if ( ARGV[i].NullQ() ) continue;
      for (auto iter = ARGV[i].Begin(); iter != ARGV[i].End(); iter ++ ){
        ndict.DictInsertPairRef( *iter ); 
      }
    }
    ReturnObject( ndict );
  }else{
    u_int i = 1;
    while ( i<= ARGV.Size() ){
      if ( ARGV[i].ListQ( SYMID_OF_List ) or
           ARGV[i].ListQ( SYMID_OF_Parenthesis ) or
           ARGV[i].ListQ( SYMID_OF_ExpressionList ) or 
           ARGV[i].ListQ( SYMID_OF_Dict )
      ){
        int s = ARGV[i].Size();
        ARGV.InsertRef( ARGV.Begin()+i, ARGV(i).Begin(), ARGV(i).End() );
        ARGV.Delete(i);
        i+= s;
      }else{
        i++;
      }
    }
    ARGV[0].SetSymbol(SYMID_OF_List);
  }
  ReturnNormal;
}

int ListModule::Dimensions(Object&ARGV){
  CheckShouldBeWithin(1,2);
  CheckShouldEqual(1);
  Object  dim; dim.SetList();
  INIT_SYMID_OF(Matrix);
  if ( ARGV.Size() == 1 ){
  //   if ( ARGV[1].ListQ( SYMID_OF_Matrix ) ){
  //     if ( ARGV[1].Size() != 1 or not ARGV[1][1].StringQ() ) { _Erroring("Matrix","Matrix should have a string to specify its name."); ReturnError; }
  //     Matrix*mat = EvaKernel->GetMatrix( ARGV[1][1].Key() );
  //     if ( mat == NULL ){
  //       { _Erroring("Dimensions",ARGV(1).ToString()+" does not exist."); ReturnError; }
  //     }
  //     int nd = mat->DimN();
  //     for (int i=1; i<=nd; i++)
  //       dim.PushBackNumber( mat->Dim(i) );
  //     ARGV = dim;
  //     ReturnNormal;
  //   }else{
  //     Object::Dimensions(ARGV[1],dim,-1);
  //     ARGV = dim;
  //     ReturnNormal;
  //   }
  // }else{ 
    CheckShouldBeNumber(2);
    int depth = ARGV[2].Number();
    Object::Dimensions(ARGV[1],dim,depth);
    ARGV = dim;
    ReturnNormal;
  }
  ReturnNormal;
}

int ListModule::Dict(Object&ARGV){

  //dout<<"into Dict current dict ="<< ARGV <<endl;
  Set_Context(Part){
    //DelaySet Calling  Dict(...) = 
    // the pass path is Part(..., ...) = ... => SetPart(...)  => Dict( ... )
    // so the form should be SetPart( partexpr,  rightvalue, valuepair[2]ref )
    // partexpr:    Part(a, 9)     a evaluated to be valuepair
    //dout<<"is dict SetPart context argv ="<<ARGV<<endl;
    //dout<<"into DelaySet of Part Dict current dict ="<< ARGV <<endl;
    CheckShouldEqual(3);
    CheckArgsShouldEqual(ARGV[1],2);
    Object&key = ARGV[1][2];
    EvaKernel->Evaluate( key );
    Object&right = ARGV[2];
    Object&dict = ARGV[3];
    Object pair(ObjectType::List, SYMID_OF_KeyValuePair );
    pair.PushBackRef( key );
    pair.PushBackCopy( ARGV[2] );
    dict.DictInsertOrUpdatePairRef( pair );
    //dout<<"new dict ="<<dict<<endl;
    //ARGV = ARGV[2];
    ReturnNull;
  }
  Part_Context(Dict){
    //dout<<"into part context of dict"<<ARGV<<endl;
    CheckShouldEqual(2);
    Object&dict = ARGV[1];
    Object&key = ARGV[2];
    Object pair = dict.DictGetPair(key);
    if ( pair.NullQ() ){
      zhWarning("字典:下标运算","不存在键值为" + key.ToString() + "的记录") ||
        _Warning("Dict:Part","No record with key value "+key.ToString() );
      ReturnNull;
    }
    ARGV.CopyObject( pair[2] );
    //dout<<"ARGV Set to "<<ARGV<<endl;
    ReturnNormal;
  }
  Set_Context(Dict){
    //dout<<"into normal DelaySet Dict current dict ="<< ARGV <<endl;
    ReturnNormal;
  }
  DelayFunction_Context(Dict){
    //dout<<"into DelayFunction of Dict current dict ="<< ARGV <<endl;
    CheckShouldEqual(1);
    Object&dict = ARGV[0];
    Object&key = ARGV[1];
    Object pair = dict.DictGetOrNewPair( key );
    ReturnObject( pair[2] );
  }

  Conjunct_Context(Dict){//Conjunct calling
    Object&dict = ARGV[1];
    Object&oper = ARGV[2];
    Conjunct_Var_Case( size ){
      ReturnNumber( dict.Size() );
    }
    Conjunct_Case2(length, size ){
      CheckArgsShouldEqual(oper,0);
      ReturnNumber( dict.Size() );
    }
    Conjunct_Case2(has, exists ){
      CheckArgsShouldEqual(oper,1);
      EvaKernel->Evaluate( oper[1] );
      Object::iterator iter;
      if ( dict.DictGetPosition( oper[1] ,iter) == 0  )
        ReturnBoolean(true);
      ReturnBoolean(false);
    }
    Conjunct_Case(get ){
      CheckArgsShouldBeWithin(oper,1,2);
      Object::iterator iter;
      EvaKernel->Evaluate( oper[1] );
      if ( oper.Size() == 1 ){
        if ( dict.DictGetPosition( oper[1] ,iter) == 0  )
          ARGV = (*iter)[2].Copy();
        else
          ARGV.SetNull();
        ReturnNormal;
      }else{
        if ( dict.DictGetPosition( oper[1] ,iter) == 0  ){
          ARGV = (*iter)[2].Copy();
        }else{
          EvaKernel->Evaluate( oper[2] );
          ARGV = oper[2];
        }
        ReturnNormal;
      }
    }
    Conjunct_Case(delete ){
      CheckArgsShouldEqual(oper,1);
      EvaKernel->Evaluate( oper[1] );
      dict.DictDelete( oper[1] ) ;
      ReturnNull;
    }
    Conjunct_Case(insert ){
      CheckArgsShouldEqual(oper,2);
      EvaKernel->Evaluate( oper[1] );
      EvaKernel->Evaluate( oper[2] );
      oper[0].SetSymbol( SYMID_OF_KeyValuePair );
      dict.DictInsertOrUpdatePairRef( oper );
      ReturnNull;
    }
    Conjunct_Case( clean ){
      CheckArgsShouldEqual( oper, 0 );
      dict.DeleteElements();
      ARGV = dict;
      ReturnNormal;
    }
    Conjunct_Case( items ){
      CheckArgsShouldEqual( oper, 0 );
      ARGV = dict.Copy();
      ARGV[0].SetSymbol(SYMID_OF_List);
      ReturnNormal;
    }
    Object l = ( oper.ListQ() ? oper[0] : oper );
    zhErroring("列表",l.ToString() + " 不是字典方法")||
      _Erroring("List",l.ToString() + " is not a Dict method.");
    ReturnNormal;
  }
  //normal calling doing nothing
  ReturnNormal;
}


int ListModule::Append(Object&ARGV)
{
  CheckShouldEqual(2);
  Object &var = ARGV[1];
  Object &ele = ARGV[2];

  if ( not var.ListQ() ){
    { _Erroring("Append","Cannot append element to an Non-List Object."); ReturnError; }
  }

  var.PushBackCopy( ele );
  ARGV = ARGV[1];
  ReturnNormal;
}

int ListModule::AppendTo(Object&ARGV)
{
  CheckShouldEqual(2);
  Object &var = ARGV[1];
  Object &ele = ARGV[2];

  if ( var.NumberQ() or var.StringQ() ){
    { _Erroring("Append","Cannot append element to an Object of Number or String."); ReturnError; }
  }
  // if not an Symbol with value assigned to 
  if ( var.ListQ() ){ 
    EvaKernel->Evaluate( var );
    if ( not var.ListQ() ){
      { _Erroring("Append","Cannot append element to Non-List Object."); ReturnError; }
    }
    var.PushBackCopy( ele );
    ARGV = ARGV[1];
    ReturnNormal;
  }
  Object res = EvaKernel->GetValuePairRef(var);
  if ( res.NullQ() )
    { _Erroring("Append","Cannot append element to an symbol without a value."); ReturnError; }
 
  if ( res.ids() != 0 ){
    if (  EvaKernel->AttributeQ(res.ids() ,AttributeType::Protected) ){
      { _Erroring("Append",(var).ToString() +(string)" is Protected."); ReturnError; }
    }
  }
  EvaKernel->Evaluate( res[2] );

  if ( not res[2].ListQ() ){
    { _Erroring("Append","Cannot append element to Non-List Object."); ReturnError; }
  }

  res[2].PushBackCopy( ele );
  ARGV.CopyObject( res[2] );
  ReturnNormal;
}

int ListModule::Insert(Object&ARGV)
{
  CheckShouldEqual(3);
  CheckShouldBeNumber(3);
  Object &var = ARGV[1];
  Object &ele = ARGV[2];
  int pos = ARGV[3].Number();

  if ( not var.ListQ() ){
    { _Erroring("Insert","Cannot insert element to Non-List Object."); ReturnError; }
  }

  if ( pos < 0 ){
    pos = var.Size() - pos + 1;
  }
  if ( pos < 1 or pos > ( (int) var.Size() + 1) )
    { _Erroring("Insert","Insert position is out of List Range."); ReturnError; }
  
  var.InsertCopy(pos, ele);
  ARGV = ARGV[1];
  ReturnNormal;
}

int ListModule::InsertTo(Object&ARGV)
{
  CheckShouldEqual(3);
  CheckShouldBeNumber(3);
  Object &var = ARGV[1];
  Object &ele = ARGV[2];
  int pos = ARGV[3].Number();

  if ( var.NumberQ() or var.StringQ() ){
    { _Erroring("Insert","Cannot insert element to an Object of Number or String."); ReturnError; }
  }
  // if not an Symbol with value assigned to 
  if ( var.ListQ() ){ 
    EvaKernel->Evaluate( var );
    if ( not var.ListQ() ){
      { _Erroring("Insert","Cannot insert element to Non-List Object."); ReturnError; }
    }
    var.InsertCopy(pos, ele );
    ARGV = ARGV[1];
    ReturnNormal;
  }
  Object res = EvaKernel->GetValuePairRef(var);
  if ( res.NullQ() )
    { _Erroring("Insert","Cannot insert element to an symbol without a value."); ReturnError; }
 
  if ( res.ids() != 0 ){
    if (  EvaKernel->AttributeQ(res.ids() ,AttributeType::Protected) ){
      { _Erroring("Insert",(var).ToString() +(string)" is Protected."); ReturnError; }
    }
  }
  EvaKernel->Evaluate( res[2] );

  if ( not res[2].ListQ() ){
    { _Erroring("Insert","Cannot insert element to Non-List Object."); ReturnError; }
  }

  res[2].InsertCopy(pos, ele );
  ARGV.CopyObject( res[2] );
  ReturnNormal;
}


int ListModule::Delete(Object&ARGV)
{
  CheckShouldEqual(2);
  CheckShouldBeNumber(2);
  Object &var = ARGV[1];
  int pos = ARGV[2].Number();
  
  if ( not var.ListQ() ){
    { _Erroring("Delete","Cannot delete element to Non-List Object."); ReturnError; }
  }
  
  if ( pos < 0 ){
    pos = var.Size() - pos + 1;
  }
  if ( pos < 1 or pos > (int) var.Size()  )
    { _Erroring("Delete","Delete position is out of List Range."); ReturnError; }
  
  var.Delete(pos);
  ARGV = ARGV[1];
  ReturnNormal;
}

int ListModule::DeleteFrom(Object&ARGV)
{
  CheckShouldEqual(2);
  CheckShouldBeNumber(2);
  Object &var = ARGV[1];
  int pos = ARGV[2].Number();

  if ( var.NumberQ() or var.StringQ() ){
    { _Erroring("Delete","Cannot delete element to an Object of Number or String."); ReturnError; }
  }
  // if not an Symbol with value assigned to 
  if ( var.ListQ() ){ 
    EvaKernel->Evaluate( var );
    if ( not var.ListQ() ){
      { _Erroring("Delete","Cannot delete element to Non-List Object."); ReturnError; }
    }
    if ( pos < 1 or pos > (int) var.Size()  )
      { _Erroring("Delete","Delete position is out of List Range."); ReturnError; }
    var.Delete(pos);
    ARGV = ARGV[1];
    ReturnNormal;
  }

  Object res = EvaKernel->GetValuePairRef(var);
  if ( res.NullQ() )
    { _Erroring("DeleteFrom","Cannot delete element to an symbol without a value."); ReturnError; }
 
  if ( res.ids() != 0 ){
    if (  EvaKernel->AttributeQ(res.ids() ,AttributeType::Protected) ){
      { _Erroring("DeleteFrom",(var).ToString() +(string)" is Protected."); ReturnError; }
    }
  }

  EvaKernel->Evaluate( res[2] );

  if ( not res[2].ListQ() ){
    { _Erroring("DeleteFrom","Cannot delete element to Non-List Object."); ReturnError; }
  }
  
  if ( pos < 1 or pos > (int) var.Size() )
    { _Erroring("Delete","Delete position is out of List Range."); ReturnError; }
  res[2].Delete( pos );
  ARGV.CopyObject( res[2] );
  ReturnNormal;
}

int ListModule::PopBack(Object&ARGV)
{
  CheckShouldEqual(1);
  Object &var = ARGV[1];

  if ( var.NumberQ() or var.StringQ() ){
    { _Erroring("PopBack","Cannot delete element to an Object of Number or String."); ReturnError; }
  }
  // if not an Symbol with value assigned to 
  if ( var.ListQ() ){ 
    EvaKernel->Evaluate( var );
    if ( not var.ListQ() ){
      { _Erroring("PopBack","Cannot delete element to Non-List Object."); ReturnError; }
    }
    Object t = var.Last();
    var.PopBack();
    ARGV = t;
    ReturnNormal;
  }
  Object res = EvaKernel->GetValuePairRef(var);
  if ( res.NullQ() )
    { _Erroring("PopBack","Cannot delete element to an symbol without a value."); ReturnError; }
 
  if ( res.ids() != 0 ){
    if (  EvaKernel->AttributeQ(res.ids() ,AttributeType::Protected) ){
      { _Erroring("PopBack",(var).ToString() +(string)" is Protected."); ReturnError; }
    }
  }
  EvaKernel->Evaluate( res[2] );

  if ( not res[2].ListQ() ){
    { _Erroring("PopBack","Cannot delete element to Non-List Object."); ReturnError; }
  }
  
  Object t = res[2].Last();
  res[2].PopBack();
  ARGV = t;
  ReturnNormal;
}


int ListModule::Reverse(Object&ARGV){
  CheckShouldEqual(1);
  CheckShouldBeList(1);
  ARGV[1].Reverse();
  ARGV = ARGV[1];
  ReturnNormal;
}

int ListModule::Transpose(Object&ARGV){
  CheckShouldEqual(1);
  CheckShouldBeList(1);
  ARGV = ARGV[1];
  int row_size = ARGV.Size();
  if ( row_size < 1 ) ReturnNormal;
  int col_size = ARGV[1].Size();
  if ( col_size < 1 ) ReturnNormal;
  if ( row_size == 1 and col_size ==1 ) ReturnNormal;
  for ( int i=2; i<= row_size; i++ ){
    if ( ARGV[i].Size() != col_size ) ReturnNormal;
  }
  if ( row_size < col_size ){ // long to to show row
    for (auto i = row_size+1; i<= col_size;i++  ) ARGV.PushBackList(SYMID_OF_List);
    for (auto i=1; i<= row_size; i++){
      // part 1  of ref swap
      for (auto j=i+1; j<=row_size; j++){
        ARGV[i][j].SwapRef(ARGV[j][i]);
      }
      // part 2 of push back ref while pop back at the same time
      for (auto j=col_size; j>row_size; j--){
        ARGV[j].PushBackRef( ARGV[i][j] );
        ARGV[i].PopBack();
      }
    }
  }else{ // short row to long row   row_size > col_size
    for (auto i=1; i<= col_size; i++){
      // part 1  of ref swap
      for (auto j=i+1; j<=col_size; j++){
        ARGV[i][j].SwapRef(ARGV[j][i]);
      }
      // part 2 of push back ref while pop back at the same time
      for (auto j=row_size; j>col_size; j--){
        ARGV[i].PushBackRef( ARGV[j][i] );
      }
    }
    ARGV.Delete( ARGV.Begin()+col_size, ARGV.End() );

  }
  ReturnNormal;
}


int ListModule::NullQ(Object&ARGV){
  CheckShouldEqual(1);
  if ( ARGV[1].NullQ() )
    ReturnSymbol(SYMID_OF_True);
  else
    ReturnSymbol(SYMID_OF_False);
}
// int ListModule::zh_NullQ(Object&ARGV){
//   CheckShouldEqual(1);
//   if ( ARGV[1].NullQ() )
//     ReturnSymbol(SYMID_OF_真);
//   else
//     ReturnSymbol(SYMID_OF_假);
// }

int ListModule::NumberQ(Object&ARGV){
  CheckShouldEqual(1);
  if ( ARGV[1].NumberQ() )
    ReturnSymbol(SYMID_OF_True);
  else
    ReturnSymbol(SYMID_OF_False);
}
// int ListModule::zh_NumberQ(Object&ARGV){
//   CheckShouldEqual(1);
//   if ( ARGV[1].NumberQ() )
//     ReturnSymbol(SYMID_OF_真);
//   else
//     ReturnSymbol(SYMID_OF_假);
// }
int ListModule::StringQ(Object&ARGV){
  CheckShouldEqual(1);
  if ( ARGV[1].StringQ() )
    ReturnSymbol(SYMID_OF_True);
  else
    ReturnSymbol(SYMID_OF_False);
}
// int ListModule::zh_StringQ(Object&ARGV){
//   CheckShouldEqual(1);
//   if ( ARGV[1].StringQ() )
//     ReturnSymbol(SYMID_OF_真);
//   else
//     ReturnSymbol(SYMID_OF_假);
// }


int ListModule::SymbolQ(Object&ARGV){
  CheckShouldBeWithin(1,2);
  if ( not ARGV[1].SymbolQ() ) ReturnSymbol( SYMID_OF_False );
  if ( ARGV.Size() == 1 ){
    ReturnSymbol(SYMID_OF_True);
  }else{
    if ( ARGV[1] == ARGV[2] )
      ReturnSymbol( SYMID_OF_True );
    ReturnSymbol( SYMID_OF_False );
  }
}
// int ListModule::zh_SymbolQ(Object&ARGV){
//   CheckShouldBeWithin(1,2);
//   if ( not ARGV[1].SymbolQ() ) ReturnSymbol( SYMID_OF_假 );
//   if ( ARGV.Size() == 1 ){
//     ReturnSymbol(SYMID_OF_真);
//   }else{
//     if ( ARGV[1] == ARGV[2] )
//       ReturnSymbol( SYMID_OF_真 );
//     ReturnSymbol( SYMID_OF_假 );
//   }
//   ReturnNormal;
// }

int ListModule::ListQ(Object&ARGV){
  CheckShouldBeWithin(1,2);
  if ( not ARGV[1].ListQ() ) ReturnSymbol( SYMID_OF_False );
  if ( ARGV.Size() == 1 ){
    if ( ARGV[1].ListQ(SYMID_OF_List) )
      ReturnSymbol(SYMID_OF_True);
    else
      ReturnSymbol(SYMID_OF_False);
  }else{
    CheckShouldBeSymbol(2);
    if ( ARGV[1].ListQ( ARGV[2] ) )
      ReturnSymbol(SYMID_OF_True);
    else
      ReturnSymbol( SYMID_OF_False );
  }
}
// int ListModule::zh_ListQ(Object&ARGV){
//   CheckShouldBeWithin(1,2);
//   if ( not ARGV[1].ListQ() ) ReturnSymbol( SYMID_OF_假 );
//   if ( ARGV.Size() == 1 ){
//     if ( ARGV[1].ListQ(SYMID_OF_List) )
//       ReturnSymbol(SYMID_OF_真);
//     else
//       ReturnSymbol(SYMID_OF_假);
//   }else{
//     CheckShouldBeSymbol(2);
//     if ( ARGV[1].ListQ( ARGV[2] ) )
//       ReturnSymbol(SYMID_OF_真);
//     else
//       ReturnSymbol( SYMID_OF_假 );
//   }
//   ReturnNormal;
// }

int ListModule::PD_EmptyQ(Object&ARGV){
  CheckShouldEqual(1);
  if ( ARGV[1].ListQ() and ARGV[1].Empty()  ){
    ARGV.SetSymbol(SYMID_OF_True);
    ReturnNormal;
  }
  ARGV.SetSymbol(SYMID_OF_False);
  ReturnNormal;
}
// int ListModule::PD_曰空(Object&ARGV){
//   CheckShouldEqual(1);
//   if ( ARGV[1].ListQ() and ARGV[1].Empty()  ){
//     ARGV.SetSymbol(SYMID_OF_真);
//     ReturnNormal;
//   }
//   ARGV.SetSymbol(SYMID_OF_假);
//   ReturnNormal;
// }

int ListModule::PD_Length(Object&ARGV){
  CheckShouldEqual(1);
  if ( ARGV[1].ListQ() ){
    ReturnNumber( ARGV[1].Size() );
  }else if ( ARGV[1].StringQ() ){
    ReturnNumber( ARGV[1].ref_string().size() );
  }
  zhWarning("长度","尝试对非列表或字符串对象求长度.") ||
    _Warning("Length","Try to apply Length on a non-List or String Object.");
  ARGV.SetNumber( 0 );
  ReturnNormal;
}

int ListModule::Map(Object&ARGV){
  CheckShouldEqual(2);
  CheckShouldNotBeNumber(1);
  CheckShouldNotBeString(1);
  CheckShouldBeList(2);
  if ( ARGV[2].Size() < 1 ){
    ARGV = ARGV[2];
    ReturnNormal;
  }
  Object&no = ARGV[1];
  no.ForceDeepen();

  for (int i=1; i<ARGV[2].Size(); i++ ){
    Object obj; obj.CopyObject( no );
    obj.PushBackRef( ARGV[2][i] );
    ARGV[2][i] = obj;
  }
  no.PushBackCopy( ARGV[2].Last() );
  ARGV[2].Last() = no;
  ARGV = ARGV[2];
  EvaKernel->Evaluate( ARGV );
  ReturnNormal;
}

int ListModule::Apply(Object&ARGV){
  CheckShouldEqual(2);
  CheckShouldNotBeNumber(1);
  CheckShouldNotBeString(1);
  CheckShouldBeList(2);
  ARGV[2][0] = ARGV[1];
  ARGV = ARGV[2];
  EvaKernel->Evaluate( ARGV );
  ReturnNormal;
}

int ListModule::Scan(Object&ARGV){
  CheckShouldEqual(2);
  CheckShouldNotBeNumber(1);
  CheckShouldNotBeString(1);
  CheckShouldBeList(2);
  if ( ARGV[2].Size() < 1 ){
    ReturnNull;
  }
  Object&no = ARGV[1];
  no.ForceDeepen();

  for (int i=1; i<ARGV[2].Size(); i++ ){
    Object obj; obj.CopyObject( no );
    obj.PushBackRef( ARGV[2][i] );
    EvaKernel->Evaluate(obj);
  }
  no.PushBackCopy( ARGV[2].Last() );
  ARGV[2].Last() = no;
  EvaKernel->Evaluate( ARGV[2].Last() );
  ReturnNull;
}

int ListModule::Nest(Object&ARGV){
  CheckShouldEqual(3);
  CheckShouldNotBeNumber(1);
  CheckShouldNotBeString(1);
  CheckShouldBeNumber(3);
  // EvaKernel->PreEvaluate( ARGV[1] );
  //dout<<"current nest argv is "<<ARGV<<endl;
 
  int N = ARGV[3].Number();
  Object&obj = ARGV[2];
  for (auto i=0; i<N - 1; i++){
    obj.DeepenAsFirst();
    //dout<<" obj deppen to "<<obj<<endl;
    // obj[0].SetObjectValuedSymbolKeeped( ARGV[1] );
    obj[0].CopyObject( ARGV[1] );
    //dout<<" obj build to "<<obj<<endl;
    EvaKernel->Evaluate( obj );
  }
  obj.DeepenAsFirst(); obj[0] = ARGV[1];
  EvaKernel->Evaluate( obj );
  ARGV = ARGV[2];
  ReturnNormal;
}

int ListModule::PD_Boolean(Object&ARGV){
  CheckShouldEqual(1);
  auto res = ARGV[1].Boolean();
  if ( res >= 0 ) ReturnBoolean( res );
  ReturnHold;
}

int ListModule::PD_Cases(Object&argv){
  /*
    ArgsN: 2,3
    Args1: List
   */
  CheckShouldBeWithin(2,3);
  CheckShouldBeList(1);
  int maxN = -1;
  if ( argv.Size() == 3 ){
    CheckShouldBeNumber(3);
    maxN = argv[3].Number();
  }
  Object matchedList(__List__);
  Object &list = argv[1];
  Object &pat = argv[2];
  //dout<<"deal with list ="<<list<<" pat ="<<pat<<endl;
  if ( pat.PairQ( SYMID_OF_Rule ) or pat.PairQ( SYMID_OF_RuleDelayed ) ){
    Pattern::UnifyRule( pat );
    Index idx = GlobalPool.Matches.New();
    vector<MatchRecord>&pairs = GlobalPool.Matches.Get( idx );

    for (auto iter=list.Begin(); iter != list.End(); iter ++ ){
      if ( maxN == 0 ) break;
      pairs.clear();
      if ( Pattern::UnifiedMatchQ(*iter,pat[1],pairs) ) { // apply change
        Object nobj = pat[2].Copy();
        Pattern::UnifiedApplyPatternPairs( nobj, pairs );
        EvaKernel->Evaluate( nobj );
        matchedList.PushBackRef( nobj );
        maxN --;
      }
    }
    GlobalPool.Matches.FreePairs( idx );
  }else{
    for (auto iter=list.Begin(); iter != list.End(); iter ++ ){
      if ( maxN == 0 ) break;
      //dout<<" test match "<<*iter <<"  ?Matched by pat "<< pat<<endl;
      if ( Pattern::MatchQ( *iter, pat ) ){
        matchedList.PushBackRef( *iter );
        maxN --;
      }
    }
  }

  ReturnObject(matchedList);
}

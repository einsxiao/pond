#include "hip/hip_runtime.h"
#include "sys/time.h"
#include "pond.h"
#include "MPIModule.h"
#include "hipfft/hipfft.h"
#include "MatrixOperationsModule.h"

#include <fftw3-mpi.h>

using namespace std;
using namespace pond;

__DEBUG_MAIN__("input.pd");

MPIModule* pond::MatrixOperationsModule::mpiModule = NULL;

int mark_time(string info = "") {
    static timeval timemark0, timemark1;
    if (info == "") {
        gettimeofday(&timemark0, NULL);
        gettimeofday(&timemark1, NULL);
    } else {
        gettimeofday(&timemark1, NULL);
        double diff_time = (timemark1.tv_sec - timemark0.tv_sec) +
                    (timemark1.tv_usec - timemark0.tv_usec) / 1000.0 / 1000.0;
        cout << info << " : " << diff_time << endl;
        timemark0 = timemark1;
    }
    return 0;
}

MatrixOperationsModule::MatrixOperationsModule() : Module(MODULE_NAME) {
    mpiModule = (pond::MPIModule*)(EvaKernel->ModulePtr("MPI"));
}

MatrixOperationsModule::~MatrixOperationsModule() {}

int MatrixOperationsModule::PD_multiThreadsTest(Object& Argv) {
    SetDataPosition(MatrixHost);
    int N = Argv[1].Number();
    int n[3] = {N, N, N};
    // FloatComplexMatrix in_data,out_data,data,data_hat;
    ComplexMatrix in_data, out_data, data, data_hat;
    // debug data init
    mark_time();
    cout << "Init data" << endl;
    in_data.Init(3, N, N, N, MatrixHostDevice);
    out_data.Init(3, N, N, N, MatrixHostDevice);
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            for (int k = 0; k < N; k++) {
                in_data(i, j, k) = sin(i + j + k);
            }
        }
    }
    //in_data.DumpFile("in.dat");
    cout << "Init data done" << endl;
    cout << "Test multiple threads transform " << endl;
    mark_time();
    MatrixOperationsModule::FFT3D(N, N, N, in_data, out_data);
    mark_time("Multiple threads FFT forward "+ToString(EvaSettings.threadNumberPerKernel) +" threads:");
    MatrixOperationsModule::FFT3D(N, N, N, out_data, in_data, pond::MatrixOperations_FFT_BACKWARD);
    mark_time("Multiple threads FFT backward "+ToString(EvaSettings.threadNumberPerKernel) +" threads:");
    // MatrixOperationsModule::FFT(2, n, in_data, out_data);

    //cout << "transform done result stored in out.dat" << endl;
    //out_data.DumpFile("out.dat");

    ////////////////////////////////////
    //  data = in_data;
    //  data.HostToDevice();
    //  pond::SetDataPosition(MatrixDevice);
    //  FFT3D( n[0], n[1], n[2], data, data, MatrixOperations_FFT_FORWARD );
    //  data.DeviceToHost();
    //  data.DumpFile("1.dat");
    //  ////////////////////////////////////
    //  FFT3D( n[0], n[1], n[2], data, data, MatrixOperations_FFT_BACKWARD );
    //  data.DeviceToHost();
    //  data.DumpFile("2.dat");
    ReturnNull;
}


void MatrixOperationsModule::Init() {}

const int PlanType_gpu_cpu = 0;
const int PlanType_float_gpu_cpu = 1;

const int PlanType_mpi = 2;
const int PlanType_float_mpi = 3;

static bool mpiFFTInited = false;
static void mpi_fft_init() {
    if (not mpiFFTInited) {
        mpiFFTInited = true;
        fftw_mpi_init();
    }
}

template <class type>
void MPI_FFT_MatrixInit(Matrix_T<type>& mat,
                        int rank,
                        int n[],
                        int ttype,
                        int batch) {
    mpi_fft_init();
    ptrdiff_t alloc_local, local_n0, local_0_start, nn[3];
    long dist = 1;
    for (int i = 0; i < rank; i++) {
        dist *= n[i];
        nn[i] = n[i];
    }
    if (ttype == PlanType_mpi) {
        alloc_local = fftw_mpi_local_size_many(
            rank, nn, batch, dist, MPI_COMM_WORLD, &local_n0, &local_0_start);
    } else {
        alloc_local = fftwf_mpi_local_size_many(
            rank, nn, batch, dist, MPI_COMM_WORLD, &local_n0, &local_0_start);
    }
    mat.Init(1, alloc_local, MatrixHostDevice);
    // mat.ReShape(3, n1/mpiModule->rankSize, n2, n3);
}

class planRecord {
   public:
    int rank;
    int type;  //
    int batch;
    int n[3];
    hipfftHandle gpu_plan;
    fftw_plan cpu_plan_forward, cpu_plan_backward;
    fftwf_plan fcpu_plan_forward, fcpu_plan_backward;
    ComplexMatrix mat;
    FloatComplexMatrix fmat;
    planRecord(int rank, int* inn, int type, int batch = 1) {
        this->rank = rank;
        this->type = type;
        this->batch = batch;
        int stride = 1, dist = 1;
        for (int i = 0; i < rank; i++) {
            this->n[i] = inn[i];
            dist *= inn[i];
        }
        // if mpi plan
        if (type == PlanType_mpi) {  // mpi
            MPI_FFT_MatrixInit(mat, rank, n, type, batch);
            ptrdiff_t nn[3];
            for (int i = 0; i < rank; i++)
                nn[i] = n[i];
            cpu_plan_forward = fftw_mpi_plan_many_dft(
                rank, nn, batch, dist, dist, (fftw_complex*)mat.Data,
                (fftw_complex*)mat.Data, MPI_COMM_WORLD, FFTW_FORWARD,
                FFTW_ESTIMATE);
            cpu_plan_backward = fftw_mpi_plan_many_dft(
                rank, nn, batch, dist, dist, (fftw_complex*)mat.Data,
                (fftw_complex*)mat.Data, MPI_COMM_WORLD, FFTW_BACKWARD,
                FFTW_ESTIMATE);
        } else if (type == PlanType_float_mpi) {  // float mpi
            MPI_FFT_MatrixInit(fmat, rank, n, type, batch);
            ptrdiff_t nn[3];
            for (int i = 0; i < rank; i++)
                nn[i] = n[i];
            fcpu_plan_forward = fftwf_mpi_plan_many_dft(
                rank, nn, batch, dist, dist, (fftwf_complex*)fmat.Data,
                (fftwf_complex*)fmat.Data, MPI_COMM_WORLD, FFTW_FORWARD,
                FFTW_ESTIMATE);
            fcpu_plan_backward = fftwf_mpi_plan_many_dft(
                rank, nn, batch, dist, dist, (fftwf_complex*)fmat.Data,
                (fftwf_complex*)fmat.Data, MPI_COMM_WORLD, FFTW_BACKWARD,
                FFTW_ESTIMATE);
        } else if (type == PlanType_gpu_cpu) {  // gpu cpu
            mat.Init(1, batch * dist, MatrixHostDevice);
            if (pond::GetParallelMode() == ParallelModeGpu) {
                hipfftPlanMany(&gpu_plan, rank, n, n, stride, dist, n, stride,
                              dist, HIPFFT_Z2Z, batch);
                CUDA_LAST_ERROR();
            }
            fftw_plan_with_nthreads(EvaSettings.threadNumberPerKernel);
            cpu_plan_forward =
                fftw_plan_many_dft(rank, inn, batch, (fftw_complex*)mat.Data, n,
                                   stride, dist, (fftw_complex*)mat.Data, n,
                                   stride, dist, FFTW_FORWARD, FFTW_ESTIMATE);
            fftw_plan_with_nthreads(EvaSettings.threadNumberPerKernel);
            cpu_plan_backward =
                fftw_plan_many_dft(rank, inn, batch, (fftw_complex*)mat.Data, n,
                                   stride, dist, (fftw_complex*)mat.Data, n,
                                   stride, dist, FFTW_BACKWARD, FFTW_ESTIMATE);
        } else {  // float gpu cpu
            fmat.Init(1, batch * dist, MatrixHostDevice);
            if (pond::GetParallelMode() == ParallelModeGpu) {
                hipfftPlanMany(&gpu_plan, rank, n, n, stride, dist, n, stride,
                              dist, HIPFFT_C2C, batch);
                CUDA_LAST_ERROR();
            }
            fftw_plan_with_nthreads(EvaSettings.threadNumberPerKernel);
            fcpu_plan_forward = fftwf_plan_many_dft(
                rank, inn, batch, (fftwf_complex*)fmat.Data, n, stride, dist,
                (fftwf_complex*)fmat.Data, n, stride, dist, FFTW_FORWARD,
                FFTW_ESTIMATE);
            fftw_plan_with_nthreads(EvaSettings.threadNumberPerKernel);
            fcpu_plan_backward = fftwf_plan_many_dft(
                rank, inn, batch, (fftwf_complex*)fmat.Data, n, stride, dist,
                (fftwf_complex*)fmat.Data, n, stride, dist, FFTW_BACKWARD,
                FFTW_ESTIMATE);
        }
    }
    ~planRecord() {}
    bool planQ(int rank, int inn[], int type, int batch = 1) {
        if (n[0] == inn[0] and n[1] == inn[1] and n[2] == inn[2] and
            this->batch == batch and this->type == type)
            return true;
        return false;
    }
    void execute(ComplexMatrix& in_mat,
                 ComplexMatrix& out_mat,
                 MatrixOperationsType dire) {
        if (type == PlanType_gpu_cpu and
            pond::GetDataPosition() == MatrixDevice) {
            if (dire == MatrixOperations_FFT_FORWARD)
                hipfftExecZ2Z(gpu_plan, (hipfftDoubleComplex*)in_mat.DataDevice,
                             (hipfftDoubleComplex*)out_mat.DataDevice,
                             HIPFFT_FORWARD);
            else
                hipfftExecZ2Z(gpu_plan, (hipfftDoubleComplex*)in_mat.DataDevice,
                             (hipfftDoubleComplex*)out_mat.DataDevice,
                             HIPFFT_BACKWARD);
            CUDA_LAST_ERROR();
        } else {
            mat = in_mat;
            if (dire == MatrixOperations_FFT_FORWARD) {
                fftw_execute(cpu_plan_forward);
            } else {
                fftw_execute(cpu_plan_backward);
            }
            out_mat = mat;
        }
    }
    void execute(FloatComplexMatrix& in_mat,
                 FloatComplexMatrix& out_mat,
                 MatrixOperationsType dire) {
        if (type == PlanType_float_gpu_cpu and
            pond::GetDataPosition() == MatrixDevice) {
            if (dire == MatrixOperations_FFT_FORWARD)
                hipfftExecC2C(gpu_plan, (hipfftComplex*)in_mat.DataDevice,
                             (hipfftComplex*)out_mat.DataDevice, HIPFFT_FORWARD);
            else
                hipfftExecC2C(gpu_plan, (hipfftComplex*)in_mat.DataDevice,
                             (hipfftComplex*)out_mat.DataDevice, HIPFFT_BACKWARD);
            CUDA_LAST_ERROR();
        } else {
            fmat = in_mat;
            if (dire == MatrixOperations_FFT_FORWARD) {
                fftwf_execute(fcpu_plan_forward);
            } else {
                fftwf_execute(fcpu_plan_backward);
            }
            out_mat = fmat;
        }
    }
    void execute(Matrix& in_mat, ComplexMatrix& out_mat) {
        if (type == PlanType_gpu_cpu and
            pond::GetDataPosition() == MatrixDevice) {
            out_mat = in_mat;
            hipfftExecZ2Z(gpu_plan, (hipfftDoubleComplex*)out_mat.DataDevice,
                         (hipfftDoubleComplex*)out_mat.DataDevice,
                         HIPFFT_FORWARD);
            CUDA_LAST_ERROR();
        } else {
            mat = in_mat;
            fftw_execute(cpu_plan_forward);
            out_mat = mat;
        }
    }
    void execute(FloatMatrix& in_mat, FloatComplexMatrix& out_mat) {
        if (type == PlanType_float_gpu_cpu and
            pond::GetDataPosition() == MatrixDevice) {
            out_mat = in_mat;
            hipfftExecC2C(gpu_plan, (hipfftComplex*)out_mat.DataDevice,
                         (hipfftComplex*)out_mat.DataDevice, HIPFFT_FORWARD);
            CUDA_LAST_ERROR();
        } else {
            fmat = in_mat;
            fftwf_execute(fcpu_plan_forward);
            out_mat = fmat;
        }
    }
    void execute(ComplexMatrix& in_mat, Matrix& out_mat) {
        if (type == PlanType_gpu_cpu and
            pond::GetDataPosition() == MatrixDevice) {
            if (not mat.SameDimensionQ(in_mat))
                mat.Init(in_mat);
            hipfftExecZ2Z(gpu_plan, (hipfftDoubleComplex*)in_mat.DataDevice,
                         (hipfftDoubleComplex*)mat.DataDevice, HIPFFT_BACKWARD);
            out_mat = mat;
            CUDA_LAST_ERROR();
        } else {
            mat = in_mat;
            fftw_execute(cpu_plan_backward);
            out_mat = mat;
        }
    }
    void execute(FloatComplexMatrix& in_mat, FloatMatrix& out_mat) {
        if (type == PlanType_gpu_cpu and
            pond::GetDataPosition() == MatrixDevice) {
            if (not fmat.SameDimensionQ(in_mat))
                fmat.Init(in_mat);
            hipfftExecC2C(gpu_plan, (hipfftComplex*)in_mat.DataDevice,
                         (hipfftComplex*)fmat.DataDevice, HIPFFT_BACKWARD);
            out_mat = fmat;
            CUDA_LAST_ERROR();
        } else {
            fmat = in_mat;
            fftwf_execute(fcpu_plan_backward);
            out_mat = fmat;
        }
    }
};
static int planTablePtr;
static vector<planRecord*> planTable;

planRecord* getPlanReady(int rank, int n[], int type, int batch = 1) {
    if (type == PlanType_mpi or type == PlanType_float_mpi) {
        mpi_fft_init();
    }
    bool found = false;
    planRecord* plan;
    // try find in table
    if (planTable.size() > 0) {
        if (planTable[planTablePtr]->planQ(rank, n, type, batch)) {
            found = true;
        } else {
            planTablePtr = 0;
            while (true) {
                if (planTable[planTablePtr]->planQ(rank, n, type, batch)) {
                    found = true;
                    break;
                }
                planTablePtr++;
                if (planTablePtr == (int)planTable.size())
                    break;
            }
        }
    }
    // not find
    if (not found) {
        static bool fftw_threads_initialized = false;
        if (not fftw_threads_initialized &&
            pond::GetParallelMode() == ParallelModeCpu) {
            fftw_init_threads();
            fftw_threads_initialized = true;
        }
        plan = new planRecord(rank, n, type, batch);
        planTable.push_back(plan);
        return plan;
    } else {
        return planTable[planTablePtr];
    }
}

void MatrixOperationsModule::FFT(int rank,
                                 int n[],
                                 ComplexMatrix& in_mat,
                                 ComplexMatrix& out_mat,
                                 MatrixOperationsType direction) {
    if (not in_mat.SameDimensionQ(out_mat)) {
        ThrowError(
            "FFT3D",
            "Input matrix and output matrix should have same dimension.");
    }
    planRecord* plan;
    int batch, product = 1;
    for (int i = 0; i < rank; i++)
        product *= n[i];
    batch = in_mat.Size() / product;
    if (batch * product != in_mat.Size()) {
        ThrowError("FFT3D",
                   "Input matrix is not consistent with transformation size.");
    }
    plan = getPlanReady(rank, n, PlanType_gpu_cpu, batch);
    plan->execute(in_mat, out_mat, direction);
    return;
}

void MatrixOperationsModule::FFT(int rank,
                                 int n[],
                                 FloatComplexMatrix& in_mat,
                                 FloatComplexMatrix& out_mat,
                                 MatrixOperationsType direction) {
    if (not in_mat.SameDimensionQ(out_mat)) {
        ThrowError(
            "FFT3D",
            "Input matrix and output matrix should have same dimension.");
    }
    planRecord* plan;
    int batch, product = 1;
    for (int i = 0; i < rank; i++)
        product *= n[i];
    batch = in_mat.Size() / product;
    if (batch * product != in_mat.Size()) {
        ThrowError("FFT3D",
                   "Input matrix is not consistent with transformation size.");
    }
    plan = getPlanReady(rank, n, PlanType_float_gpu_cpu, batch);
    plan->execute(in_mat, out_mat, direction);
    return;
}

void MatrixOperationsModule::FFT3D(int n1,
                                   int n2,
                                   int n3,
                                   ComplexMatrix& in_mat,
                                   ComplexMatrix& out_mat,
                                   MatrixOperationsType direction) {
    if (not in_mat.SameDimensionQ(out_mat)) {
        ThrowError(
            "FFT3D",
            "Input matrix and output matrix should have same dimension.");
    }
    planRecord* plan;
    int n[3] = {n1, n2, n3}, batch;
    batch = in_mat.Size() / (n1 * n2 * n3);
    if (batch * n1 * n2 * n3 != in_mat.Size()) {
        ThrowError("FFT3D",
                   "Input matrix is not consistent with transformation size.");
    }
    plan = getPlanReady(3, n, PlanType_gpu_cpu, batch);
    plan->execute(in_mat, out_mat, direction);
    return;
}

void MatrixOperationsModule::FFT3D(int n1,
                                   int n2,
                                   int n3,
                                   FloatComplexMatrix& in_mat,
                                   FloatComplexMatrix& out_mat,
                                   MatrixOperationsType direction) {
    if (not in_mat.SameDimensionQ(out_mat))
        ThrowError(
            "FFT3D",
            "Input matrix and output matrix should have same dimension.");
    planRecord* plan;
    int n[3] = {n1, n2, n3}, batch;
    batch = in_mat.Size() / (n1 * n2 * n3);
    if (batch * n1 * n2 * n3 != in_mat.Size())
        ThrowError("FFT3D",
                   "Input matrix is not consistent with transformation size.");
    plan = getPlanReady(3, n, PlanType_float_gpu_cpu, batch);
    plan->execute(in_mat, out_mat, direction);
    return;
}

void MatrixOperationsModule::FFT3D(int n1,
                                   int n2,
                                   int n3,
                                   Matrix& in_mat,
                                   ComplexMatrix& out_mat) {
    if (not in_mat.SameDimensionQ(out_mat))
        ThrowError(
            "FFT3D",
            "Input matrix and output matrix should have same dimension.");
    planRecord* plan;
    int n[3] = {n1, n2, n3}, batch;
    batch = in_mat.Size() / (n1 * n2 * n3);
    if (batch * n1 * n2 * n3 != in_mat.Size())
        ThrowError("FFT3D",
                   "Input matrix is not consistent with transformation size.");
    plan = getPlanReady(3, n, PlanType_gpu_cpu, batch);
    plan->execute(in_mat, out_mat);
    return;
}

void MatrixOperationsModule::FFT3D(int n1,
                                   int n2,
                                   int n3,
                                   FloatMatrix& in_mat,
                                   FloatComplexMatrix& out_mat) {
    if (not in_mat.SameDimensionQ(out_mat))
        ThrowError(
            "FFT3D",
            "Input matrix and output matrix should have same dimension.");
    planRecord* plan;
    int n[3] = {n1, n2, n3}, batch;
    batch = in_mat.Size() / (n1 * n2 * n3);
    if (batch * n1 * n2 * n3 != in_mat.Size())
        ThrowError("FFT3D",
                   "Input matrix is not consistent with transformation size.");
    plan = getPlanReady(3, n, PlanType_float_gpu_cpu, batch);
    plan->execute(in_mat, out_mat);
    return;
}

void MatrixOperationsModule::FFT3D(int n1,
                                   int n2,
                                   int n3,
                                   ComplexMatrix& in_mat,
                                   Matrix& out_mat) {
    if (not in_mat.SameDimensionQ(out_mat))
        ThrowError(
            "FFT3D",
            "Input matrix and output matrix should have same dimension.");
    planRecord* plan;
    int n[3] = {n1, n2, n3}, batch;
    batch = in_mat.Size() / (n1 * n2 * n3);
    if (batch * n1 * n2 * n3 != in_mat.Size())
        ThrowError("FFT3D",
                   "Input matrix is not consistent with transformation size.");
    plan = getPlanReady(3, n, PlanType_gpu_cpu, batch);
    plan->execute(in_mat, out_mat);
    return;
}

void MatrixOperationsModule::FFT3D(int n1,
                                   int n2,
                                   int n3,
                                   FloatComplexMatrix& in_mat,
                                   FloatMatrix& out_mat) {
    if (not in_mat.SameDimensionQ(out_mat))
        ThrowError(
            "FFT3D",
            "Input matrix and output matrix should have same dimension.");
    planRecord* plan;
    int n[3] = {n1, n2, n3}, batch;
    batch = in_mat.Size() / (n1 * n2 * n3);
    if (batch * n1 * n2 * n3 != in_mat.Size())
        ThrowError("FFT3D",
                   "Input matrix is not consistent with transformation size.");
    plan = getPlanReady(3, n, PlanType_float_gpu_cpu, batch);
    plan->execute(in_mat, out_mat);
    return;
}

void MatrixOperationsModule::MPI_FFT3D(int n1,
                                       int n2,
                                       int n3,
                                       ComplexMatrix& in_mat,
                                       ComplexMatrix& out_mat,
                                       MatrixOperationsType direction) {
    if (not in_mat.SameDimensionQ(out_mat))
        ThrowError(
            "FFT3D",
            "Input matrix and output matrix should have same dimension.");
    if ((mpiModule->rankSize * in_mat.Size()) != (n1 * n2 * n3))
        ThrowError("FFT3D", "Matrix size does not match transformation size.");
    if ((mpiModule->rankSize * in_mat.D1) != n1)
        ThrowError("FFT3D",
                   "Divided Matrix size of Dimension 1st does not match "
                   "transformation size.");
    mpi_fft_init();
    planRecord* plan;
    int n[3] = {n1, n2, n3};
    plan = getPlanReady(3, n, PlanType_mpi, 1);
    if (pond::GetDataPosition() == MatrixHost) {
        plan->execute(in_mat, out_mat, direction);
        out_mat.ReShape(3, n1 / mpiModule->rankSize, n2, n3);
    } else {
        cout << "acc plan apply in Device. Code not ready do nothing.\n"
             << endl;
    }
    return;
}

void MatrixOperationsModule::MPI_FFT3D(int n1,
                                       int n2,
                                       int n3,
                                       Matrix& in_mat,
                                       ComplexMatrix& out_mat) {
    if (not in_mat.SameDimensionQ(out_mat))
        ThrowError(
            "FFT3D",
            "Input matrix and output matrix should have same dimension.");
    if ((mpiModule->rankSize * in_mat.Size()) != (n1 * n2 * n3))
        ThrowError("FFT3D", "Matrix size does not match transformation size.");
    if ((mpiModule->rankSize * in_mat.D1) != n1)
        ThrowError("FFT3D",
                   "Divided Matrix size of Dimension 1st does not match "
                   "transformation size.");
    mpi_fft_init();
    planRecord* plan;
    int n[3] = {n1, n2, n3};
    plan = getPlanReady(3, n, PlanType_mpi, 1);
    plan->execute(in_mat, out_mat);
    out_mat.ReShape(3, n1 / mpiModule->rankSize, n2, n3);
    return;
}

int MatrixOperationsModule::PD_MatrixMPI_FFT3DC2C(Object& argv) {
    ReturnNull;
}

#include "hip/hip_runtime.h"
//////////////////////////////////////////////
#include"pond.h"
#include"SystemModule.h"
#include"ListModule.h"
#include<sys/time.h>
//////////////////////////////////////////////
using namespace std;
using namespace pond;

static INIT_SYMID_OF( SimpleSimplify );
static INIT_SYMID_OF( Do );
static INIT_SYMID_OF( Attributes );
static INIT_SYMID_OF( FUNCTION$DEFINED$ );
// static INIT_OPER_SYMID_OF( 如果,        0,220,   1,2    );
// static INIT_OPER_SYMID_OF( 又如,      218,219,   3,2    );
// static INIT_OPER_SYMID_OF( 否则,      218,219,   3,1    );
// static INIT_OPER_SYMID_OF( 只要,        0,220,   1,2    );
// static INIT_OPER_SYMID_OF( 罗列,        0,220,   1,2    );
// static INIT_OPER_SYMID_OF( 于  ,      120,115,   3,1    );

static INIT_OPER_SYMID_OF( def ,        0,220,   1,2    );
static INIT_OPER_SYMID_OF( function ,   0,220,   1,2    );
// static INIT_OPER_SYMID_OF( 函数,        0,220,   1,2    );
// static INIT_OPER_SYMID_OF( 存于,      120,115,   3,1    );
static INIT_OPER_SYMID_OF( in  ,      120,115,   3,1    );
// static INIT_OPER_SYMID_OF( 为  ,      120,115,   3,1    );
static INIT_OPER_SYMID_OF( is  ,      120,115,   3,1    );
static INIT_OPER_SYMID_OF( do  ,        0,220,   1,2    );

static INIT_OPER_SYMID_OF( context,     0,220,   1,1    );
static INIT_OPER_SYMID_OF( ctx,         0,220,   1,1    );
    
__DEBUG_MAIN__("input.pd");

SystemModule::SystemModule():Module(MODULE_NAME){
  //dout<<"system module initialized"<<endl;

  ///////////////////////////////////////////////////////////
  // special form
#define AP(sym) EvaKernel->SetAttribute(#sym, AttributeType::Protected )
  AP(Null);
  AP(Flat);
  AP(Listable);
  AP(OneIdentity);
  AP(HoldFirst);
  AP(HoldRest);
  AP(HoldAll);
  AP(HoldAllCompleted);
  AP(SequenceHold);
  AP(Orderless);
  AP(GrammarUnit);
  AP(Constant);
  AP(DelayFunction);
  AP(DelayHold);
  AP(DelaySetable);

  AP(Conjunctable);
  AP(Setable);
  AP(Partable);
  AP(Destroyable);
  AP(Protected);
  AP(Locked);
  AP(MAXENUM);
#undef AP
  Object left,right;
  left.SetSymbol("I"); right.SetSymbol("I");
  EvaKernel->InsertOrUpdateBottomPairValue(left,right);
  AddAttribute("I",AttributeType::Protected );

  // systematic functions
  // {
  //   RegisterFunction("Evaluate",Evaluate,this);
  // }
  {
    RegisterFunction("Set",Set,this);
    AddAttribute("Set",AttributeType::HoldFirst);
  }
  {
    RegisterFunction("SetDelayed",          SetDelayed,this);
    AddAttribute("SetDelayed",              AttributeType::HoldAll);
  }
  {
    AddAttribute("context",  AttributeType::HoldAll );
    RegisterFunction( "ctx",MemberFunction(&SystemModule::PD_context), this);
    AddAttribute("ctx",  AttributeType::HoldAll );
  }
  {
    RegisterFunction("Parenthesis",         Parenthesis,this);
    AddAttribute("Parenthesis",  AttributeType::HoldAll );
    AddAttribute("Parenthesis",  AttributeType::OneIdentity );
  }
  {
    RegisterFunction("ExpressionList",      ExpressionList,this);
    AddAttribute("ExpressionList",  AttributeType::HoldAll );
  }
  {
    RegisterFunction("CompoundExpression",  CompoundExpression,this);
    AddAttribute("CompoundExpression",  AttributeType::HoldAll );
  }
  {
    RegisterFunction("Replace",             Replace,this);
    AddAttribute("Replace",  AttributeType::HoldAll );
  }
  {
    RegisterFunction("ReplaceAll",          ReplaceAll,this);
    AddAttribute("ReplaceAll",  AttributeType::HoldAll );
  }
  {
    RegisterFunction("ReplaceAllRepeated",  ReplaceAllRepeated,this);
    AddAttribute("ReplaceAllRepeated",  AttributeType::HoldAll );
  }
  {
    RegisterFunction("ShowModuleFunctions",ShowModuleFunctions,this);
    AddAttribute("ShowModuleFunctions",AttributeType::Listable);
  }
  {
    RegisterFunction("ShowFunctionsOfModule",ShowModuleFunctions,this);
    AddAttribute("ShowFunctionsOfModule",AttributeType::Listable);
  }
  {
    RegisterFunction("ShowModules",ShowModules,this);
  }
  ////////////////////////////////////////////////////////////////
  {
    // RegisterFunction("Options",Options,this);
  }
  {
    // RegisterFunction("Attributes",Attributes,this); 
    AddAttribute("Attributes",AttributeType::Listable);
  }
  {
    RegisterFunction("SetAttributes",SetAttribute_Eva,this);
    AddAttribute("SetAttributes",AttributeType::HoldAll);
    AddAttribute("Attributes",AttributeType::Setable);
  }
  {
    RegisterFunction("AddAttribute",AddAttribute_Eva,this);
    AddAttribute("AddAttribute",AttributeType::HoldAll);
  }
  ///////////////////////////////////////////////////////////
  ///////////////////////////////////////////////////////////
  // pattern system
  {
    RegisterFunction("MatchQ",MatchQ,this);
  }
  {
    RegisterFunction("UnifyRule",UnifyRule,this);
  }
  {
    RegisterFunction("UnifiedMatchQ",UnifiedMatchQ,this);
  }
  ////////////////////////////////////////////////////////////
  {
    AddAttribute("Pattern",AttributeType::HoldFirst);
    AddAttribute("Pattern",AttributeType::Protected);
  }
  {
    AddAttribute("Rule",AttributeType::Protected);
    AddAttribute("RuleDelayed",AttributeType::HoldRest);
    AddAttribute("RuleDelayed",AttributeType::Protected);
    AddAttribute("KeyValuePair",AttributeType::HoldFirst);
    AddAttribute("KeyValuePair",AttributeType::Protected);
  }
  ////////////////////////////////////////////////////////////
  {
    // RegisterFunction("Hold",Hold,this);
    AddAttribute("Hold",AttributeType::HoldAll);
    AddAttribute("Hold",AttributeType::OneIdentity);
    AddAttribute("Hold",AttributeType::OneIdentity);

  }
  {
    RegisterFunction("HoldOnce",HoldOnce,this);
    AddAttribute("HoldOnce",AttributeType::HoldAll);
    AddAttribute("HoldOnce",AttributeType::OneIdentity);
  }
  {
    // RegisterFunction("HoldPattern",HoldPattern,this);
    AddAttribute("HoldPattern",AttributeType::HoldAll);
    AddAttribute("HoldPattern",AttributeType::OneIdentity);
  }
  ///////////////////////////////////////////////////////////
  {
    RegisterFunction("Module",GrammarModule,this);
    AddAttribute("Module",AttributeType::HoldAll);
  }
  {
    RegisterFunction("Function",Function,this);
    AddAttribute("Function",AttributeType::DelayFunction);
    AddAttribute("Function",AttributeType::HoldAll);

    RegisterFunction("PureFunction",PureFunction,this);
    AddAttribute("PureFunction",AttributeType::DelayFunction);
    AddAttribute("PureFunction",AttributeType::HoldAll);

    // AddAttribute("ArrowFunction",AttributeType::DelayFunction);
    AddAttribute("ArrowFunction",AttributeType::HoldAll); // working similar to def and function

    RegisterFunction( "function",MemberFunction(&SystemModule::PD_def), this);
    AddAttribute("def",AttributeType::HoldAll);
    AddAttribute("function",AttributeType::HoldAll);
    
    RegisterFunction( "函数",MemberFunction(&SystemModule::PD_def), this);
    AddAttribute("函数",AttributeType::HoldAll);
    AddAttribute("FUNCTION$DEFINED$",AttributeType::DelayFunction);
    AddAttribute("FUNCTION$DEFINED$",AttributeType::HoldAll);

  }
  {
    RegisterFunction("Conjunct",Conjunct,this);
    AddAttribute("Conjunct",AttributeType::HoldAll );
  }
  ///////////////////////////////////////////////////////////
  {
    RegisterFunction("If",If,this);
    AddAttribute("If",AttributeType::HoldAll);
  }
  {
    RegisterFunction("While",While,this);
    AddAttribute("While",AttributeType::HoldAll);
  }
  {
    RegisterFunction("For",For,this);
    AddAttribute("For",AttributeType::HoldAll);
  }
  {
    RegisterFunction("Foreach",Foreach,this);
    AddAttribute("Foreach",AttributeType::HoldAll);
  }
  {
    RegisterFunction("Do",Do,this);
    AddAttribute("Do",AttributeType::HoldAll);
    AddAttribute("do",AttributeType::HoldAll);
  }
  {
     
    
    RegisterFunction("如果", (MemberFunction)&SystemModule::PD_if, this );
    AddAttribute("如果",  AttributeType::HoldAll );
    AddAttribute("又如",AttributeType::Protected  );
    AddAttribute("否则",AttributeType::Protected );
    AddAttribute("if",  AttributeType::HoldAll );
    AddAttribute("elif",AttributeType::Protected );
    AddAttribute("else",AttributeType::Protected );
    RegisterFunction("只要", (MemberFunction)&SystemModule::PD_while, this );
    AddAttribute("只要",AttributeType::HoldAll);
    AddAttribute("while",AttributeType::HoldAll);
    RegisterFunction("罗列", (MemberFunction)&SystemModule::PD_for, this );
    AddAttribute("罗列",AttributeType::HoldAll);
    AddAttribute("for",AttributeType::HoldAll);
    RegisterFunction("于", (MemberFunction)&SystemModule::PD_in, this );
    AddAttribute("于",AttributeType::Protected );
    AddAttribute("in",AttributeType::Protected );
    AddAttribute("do",AttributeType::HoldAll);
    AddAttribute("switch",AttributeType::HoldAll);
    AddAttribute("case",AttributeType::Protected );
    AddAttribute("default",AttributeType::Protected );
    AddAttribute("in",AttributeType::HoldAll);
    AddAttribute("var",AttributeType::HoldAll);
  }
  // {
  //   RegisterFunction("System",System,this);
  //   RegisterFunction("SystemWithoutOutput",SystemWithoutOutput,this);
  // }
  ///////////////////////////////////////////////////////////
  RegisterFunction("SetOutputFormat",SetOutputFormat,this,"Set output format for numbers.\nSetOutputFormat[precison,is_scientific]"); 
  RegisterFunction("SystemArgv",SystemArgv,this,"Get system input argument variables.");
  ///////////////////////////////////////////////////////////
  RegisterFunction("Help",ShowHelpInfo,this,"Show help information of module functions.");
  RegisterFunction("Clear",Clear,this,"Clear defination relating to symbol(s)."); 
  AddAttribute("Clear",AttributeType::HoldAll);

  RegisterFunction("clear",Clear,this,"Clear defination relating to symbol(s)."); 
  AddAttribute("clear",AttributeType::HoldAll);

  //RegisterFunction("Protect",Protect_Eva,this,"Add Protect Attribute to a Symbol.");
  AddAttribute("Protect",AttributeType::HoldAll);
  //RegisterFunction("UnProtect",UnProtect_Eva,this,"Remove Protect Attribute of a Symbol if exist.");
  AddAttribute("UnProtect",AttributeType::HoldAll);
  ///////////////////////////////////////////////////////////
  RegisterFunction("AbsoluteTime",AbsoluteTime,this,"Get Absolute time from 1970.");
  RegisterFunction("DateList",DateList,this,"Give out the Date Object .");
  {
    RegisterFunction("Timing",Timing,this,"Calculate the time a command consumed."); 
    AddAttribute("Timing",AttributeType::HoldAll);
  }
  {
    RegisterFunction("CpuTiming",CpuTiming,this,"Calculate the Cpu time a command consumed."); 
    AddAttribute("CpuTiming",AttributeType::HoldAll);
  }
  RegisterFunction("Sleep",Sleep,this,"Sleep for some seconds. Then continue to run.");
  RegisterFunction("sleep",Sleep,this,"Sleep for some seconds. Then continue to run.");

  RegisterFunction("ToString",ToString_Eva,this);
  RegisterFunction("string",ToString_Eva,this);
  RegisterFunction("ToExpression",ToExpression,this);

  {
    RegisterFunction("SimpleSimplify", SimpleSimplify, this );
  }
  AddAttribute("Profile",AttributeType::HoldAll);

  //dout<<"System register funtion list = "<< __module_function_list <<" id="<<__module_function_list.objid<<endl;
  AddAttribute("type",   AttributeType::HoldAll);
  AddAttribute("String", AttributeType::Protected);
  AddAttribute("Number", AttributeType::Protected);
  AddAttribute("Symbol", AttributeType::Protected);
  AddAttribute("List",   AttributeType::Protected);

};

SystemModule::~SystemModule(){
};

int SystemModule::PD_Evaluate(Object &ARGV){
  /*zh:
    Evaluate(expr)
        对处于Hold状态中的表达式expr进行求值
    |||
    Evaluate(expr)
        Evaluate expr in 'Hold' state
   */
  CheckShouldEqual(1);
  EvaKernel->Evaluate(ARGV[1],false);
  ARGV = ARGV[1];
  ReturnNormal;
}

int SystemModule::PD_EvaluateString(Object &ARGV){
  /*zh:
    EvaluateString(str)
        解析str中的表达式并求值
    |||
    EvaluateString(str)
        parse expression in str and evaluate the parsed result
   */
  CheckShouldEqual(1);
  CheckShouldBeString(1);
  string code = ARGV[1].Key(); 
  pond::PondInnerStringRestoreNormal( code );
  Object result;
  EvaKernel->EvaluateString(code, result, 0, true);
  ARGV = result;
  ReturnObject(result);
}

int SystemModule::PD_EvaluateFile(Object &ARGV){
  /*zh:
    EvaluateFile(filename)
        解析文件中的内容并求值
    |||
    EvaluateFile(filename)
        parse content in file and evaluate the parsed result
   */
  CheckShouldEqual(1);
  CheckShouldBeString(1);
  Object result;
  EvaKernel->EvaluateFile(ARGV[1].Key(), result, 0, false);
  ReturnNull;
}

int SystemModule::PD_EvaluateFileWithReturn(Object &ARGV){
  /*zh:
    EvaluateFile(filename)
        解析文件中的内容并求值
    |||
    EvaluateFile(filename)
        parse content in file and evaluate the parsed result
   */
  CheckShouldEqual(1);
  CheckShouldBeString(1);
  Object result;
  EvaKernel->EvaluateFile(ARGV[1].Key(), result, 0, false);
  ReturnObject(result);
}

int SystemModule::PD_Options(Object & ARGV){
  ReturnHold;
}

int SystemModule::SetOptions(Object & ARGV){
  ReturnHold;
}

int SystemModule::SetAttribute(Object &left,Object&attris){
  if ( not attris.ListQ(SYMID_OF_List) ) {
    zhErroring("设置特性", "特性参数应当以列表的形式传入.") ||
    _Erroring("SetAttribute", "Attributes should be in a List.");
    ReturnError;
  }
  for ( u_int i =1 ;i<= attris.Size();i++){
    if (not(attris)[i].StringQ())
    {
      zhErroring( "设置属性", "属性应该用字符串表示") ||
        _Erroring("SetAttibute", "Attibutes should be Strings.");
      ReturnError;
    }
  }
  bool *attri;
  if ( left.ListQ( SYMID_OF_Attributes) ){
    if ((left).Size() != 1 ) {
      zhErroring("SetAttributes","Attributes 要求以Object作为参数.")||
        _Erroring("SetAttributes","Attributes requires an Object as argument.");
      ReturnError; }
    if ( not left[1].SymbolQ() ) {
      zhErroring("SetAttribute","只有符号变量才能被赋予属性.")||
        _Erroring("SetAttribute","Only Symbol can assign attributes to.");
      ReturnError;
    }
    attri = EvaKernel->GetAttributes( (left)[1].Key() );
  }else{
    if ( not left.SymbolQ() ) {
      zhErroring("SetAttribute","只有符号变量才能被赋予属性.")||
        _Erroring("SetAttribute","Only Symbol can assign attributes to.");
      ReturnError;
    }
    attri = EvaKernel->GetAttributes( left );
  }
  if ( attri == NULL ) {
    zhErroring("SetAttributes","不能创建新的属性.")||
      _Erroring("SetAttributes","Can not create new attributes.");
    ReturnError;
  }
  
  AttributeType type;
  for (u_int i=1; i<=attris.Size(); i++ ){
    if ( not attris[i].StringQ() ) {
      zhErroring("SetAttributes","要求一个属性字符串.")||
        _Erroring("SetAttributes","An attribute string is required.");
      ReturnError;
    }
    type = String2AttributeType( attris[i].Key() );
    if ( type == AttributeType::Null ) {
      zhErroring("SetAttribute",(string)"未知属性 \""+attris[i].Key()+"\".")||
        _Erroring("SetAttribute",(string)"Unkonwn Attribute \""+attris[i].Key()+"\".");
      ReturnError;
    }
    pond::SetAttribute(attri,type);
  }
  return 0;
};

int SystemModule::SetAttribute_Eva(Object &ARGV){
  CheckArgsShouldEqual(ARGV,2);
  SetAttribute( ARGV[1], ARGV[2]);
  ARGV = ARGV[2];
  ReturnNormal;
};

int SystemModule::AddAttribute_Eva(Object &ARGV){
  CheckShouldEqual(2);
  CheckShouldBeSymbol(1);
  EvaRecord *rec = EvaKernel->GetOrNewEvaRecord( ARGV[1] );
  if ( rec == NULL ) {
    zhErroring("AddAttribute","未能获取符号变量 '"+ARGV[1].ToString()+"' 的参数对象.")||
      _Erroring("AddAttribute","Can not get Attributes Object for Symbol '"+ARGV[1].ToString()+"'.");
    ReturnError;
  }
  if ( AttributeQ(rec->attributes,AttributeType::Protected) ) {
    zhErroring("AddAttribute",(string)"符号变量'"+ARGV[1].Key()+"' 处于保护状态中.")||
      _Erroring("AddAttribute",(string)"Symbol '"+ARGV[1].Key()+"' is protected.");
    ReturnError;
  }
  if ( ARGV[2].SymbolQ() || ARGV[2].StringQ() ){
    AttributeType attriType = String2AttributeType( ARGV[2].Key() );
    if ( attriType == AttributeType::Null ) {
      zhErroring("AddAttribute",(string)"给出的属性类型'"+ARGV[2].Key()+"' 不正确.")||
        _Erroring("AddAttribute",(string)"Attribute type '"+ARGV[2].Key()+"' specified is not right.");
      ReturnError;
    }
    pond::SetAttribute(rec->attributes, attriType );
    ReturnNull;
  } else if ( ARGV[2].ListQ(SYMID_OF_List) ){
    AttributeType attriType;
    for ( u_int i =1; i<=ARGV[2].Size() ; i++){
      attriType = String2AttributeType( ARGV[2][i].Key() );
      if ( attriType == AttributeType::Null ) {
        zhErroring("AddAttribute",(string)"给出的属性类型'"+ARGV[2][i].Key()+"' 不正确.")||
          _Erroring("AddAttribute",(string)"Attribute type '"+ARGV[2][i].Key()+"' specified is not right.");
        ReturnError;
      }
      pond::SetAttribute( rec->attributes, attriType );
    }
    ReturnNull;
  }
  zhErroring("AddAttribute","给出的属性值形式不正确.")||
    _Erroring("AddAttribute","Attributes specified is not in the right form.");
  ReturnError;
};

int SystemModule::RemoveAttributes_Eva(Object &ARGV){
  CheckShouldEqual(2);
  CheckShouldBeSymbol(1);
  EvaRecord *rec = EvaKernel->GetOrNewEvaRecord( ARGV[1] );
  if ( rec == NULL ) {
    zhErroring("RemoveAttribute","未能获取符号变量 '"+ARGV[1].ToString()+"' 的参数对象.")||
      _Erroring("RemoveAttribute","Can not get Attributes Object for Symbol '"+ARGV[1].ToString()+"'.");
    ReturnError;
  }
  if ( AttributeQ(rec->attributes,AttributeType::Protected) ) {
    zhErroring("RemoveAttribute",(string)"符号变量'"+ARGV[1].Key()+"' 处于保护状态中.")||
      _Erroring("RemoveAttribute",(string)"Symbol '"+ARGV[1].Key()+"' is protected.");
    ReturnError;
  }
  if ( ARGV[2].SymbolQ() || ARGV[2].StringQ() ){
    AttributeType attriType = String2AttributeType( ARGV[2].Key() );
    if ( attriType == AttributeType::Null ) {
      zhErroring("RemoveAttribute",(string)"给出的属性类型'"+ARGV[2].Key()+"' 不正确.")||
        _Erroring("RemoveAttribute",(string)"Attribute type '"+ARGV[2].Key()+"' specified is not right.");
      ReturnError;
    }
    pond::SetAttribute( rec->attributes, attriType,false);
    ReturnNull;
  }else if ( ARGV[2].ListQ(SYMID_OF_List) ){
    AttributeType attriType;
    for ( u_int i =1; i<=ARGV[2].Size() ; i++){
      attriType = String2AttributeType( ARGV[2][i].Key() );
      if ( attriType == AttributeType::Null ) {
        zhErroring("RemoveAttribute",(string)"给出的属性类型'"+ARGV[2][i].Key()+"' 不正确.")||
          _Erroring("RemoveAttribute",(string)"Attribute type '"+ARGV[2][i].Key()+"' specified is not right.");
        ReturnError;
      }
      pond::SetAttribute( rec->attributes, attriType,false );
    }
    ReturnNull;
  }
  zhErroring("RemoveAttribute","给出的属性值形式不正确.")||
    _Erroring("RemoveAttribute","Attributes specified is not in the right form.");
  ReturnError;
};

int SystemModule::PD_Protect(Object &ARGV){
  /*zh:
    Protect( [sym] )
        为[sym]添加保护属性(Protected)，禁止对[sym]进行修改操作
    |||
    Protect( [sym] )
        Add a 'Protected' attribute to symbol [sym], prevent any change to [sym]
  */
     
  CheckShouldEqual(1);
  CheckShouldBeSymbol(1);
  EvaRecord *rec = EvaKernel->GetOrNewEvaRecord( ARGV[1] );
  if ( rec == NULL ) {
    zhErroring("Protect","不能获得符号'"+ARGV[1].ToString()+"'的属性对象")||
      _Erroring("Protect","Can not get Attributes Object for Symbol '"+ARGV[1].ToString()+"'");
    ReturnError;
  }
  pond::SetAttribute( rec->attributes, AttributeType::Protected, true);
  ReturnNull;
};

int SystemModule::PD_UnProtect(Object &ARGV){
  /*zh:
    UnProtect( [sym] )
        移除[sym]的保护属性(Protected)，恢复对[sym]进行修改操作的权限
    |||
    UnProtect( [sym] )
        Remove 'Protected' attribute of symbol [sym]
  */
  CheckShouldEqual(1);
  CheckShouldBeSymbol(1);
  EvaRecord *rec = EvaKernel->GetOrNewEvaRecord( ARGV[1] );
  if ( rec == NULL ) {
    zhErroring("移除属性","不能获得符号"+ARGV[1].ToString()+"的属性对象") ||
      _Erroring("RemoveAttribute","Can not get Attributes Object for Symbol"+ARGV[1].ToString() );
    ReturnError;
  }
  pond::SetAttribute( rec->attributes, AttributeType::Protected,false);
  ReturnNull;
};

int SystemModule::PD_Attributes(Object & ARGV){
  Set_Context(Attributes){
    return SetAttribute_Eva(ARGV);
  }
  CheckArgsShouldEqual(ARGV,1);
  CheckShouldBeSymbol(1);
  Object arg = ARGV(1);
  bool *attri = EvaKernel->GetAttributes( arg.Key() );
  ARGV.SetList();
  if ( attri ){
    for ( int i = (int)AttributeType::Null+1; i< (int)AttributeType::MAXENUM; i++ ){ 
      if ( attri[i] )
        ARGV.PushBackString( AttributeType2String((AttributeType)i) );
    }
  }
  ReturnNormal;
}

int SystemModule::Parenthesis(Object & ARGV){
  //dout<<"comes to ()"<<ARGV<<endl;
  for (int i = 1;  i <= ARGV.Size() ; i++){
    EvaKernel->Evaluate( ARGV[i] );
    //dout<<"one () run end to "<<ARGV<<endl;
    if ( EvaKernel->statusCode == 88 ){
      ARGV = EvaKernel->statusObject;
      ReturnNormal;
    }else if( EvaKernel->statusCode == 66 or EvaKernel->statusCode == 44 ){
      ReturnNull;
    }
  }
  //dout<<"() evaluated to "<<ARGV<<endl;
  if ( ARGV.Size() == 0 ){
    ReturnNull;
  }
  if ( ARGV.Size() == 1 ){
    ARGV = ARGV[ 1 ];
    ReturnNormal;
  }
  // ARGV[0].SetSymbol( SYMID_OF_Tuple );
  ReturnNormal;
}

int SystemModule::ExpressionList(Object & ARGV){
  int stackPtr = EvaKernel->GetStackPtr();
  for (int i = 1;  i <= ARGV.Size() ; i++){
    EvaKernel->Evaluate( ARGV[i] );
    //dout<<"Expression evaluate "<<i<<" to "<<ARGV<<endl;
    if ( EvaKernel->statusCode == 88 ){
      ARGV = EvaKernel->statusObject;
      EvaKernel->SetStackToPtr( stackPtr );
      ReturnNormal;
    }else if( EvaKernel->statusCode == 66 or
              EvaKernel->statusCode == 44 ){
      EvaKernel->SetStackToPtr( stackPtr );
      ReturnNull;
    }
  }
  EvaKernel->SetStackToPtr( stackPtr );
  //dout<<"Expression evaluate end to "<<ARGV<<endl;
  if ( ARGV.Size() == 0 ){
    ReturnNull;
  }
  if ( ARGV.Size() == 1 ){
    ARGV = ARGV[ 1 ];
  }
  //dout<<"Expression evaluate final end to "<<ARGV<<endl;
  ReturnNormal;
}

int SystemModule::CompoundExpression(Object & ARGV){
  for (auto iter = ARGV.Begin(); iter!= ARGV.End() ; iter++){
    EvaKernel->Evaluate(*iter);
    if ( EvaKernel->statusCode == 88 ){
      ARGV = EvaKernel->statusObject;
      ReturnNormal;
    }else if( EvaKernel->statusCode == 66 or EvaKernel->statusCode == 44 ){
      ReturnNull;
    }
  }
  //dout<<" ce = "<<ARGV<<endl;
  if ( ARGV.Size() > 0 ) 
    ARGV = ARGV[ ARGV.Size() ];
  else
    ARGV.SetNull();
  ReturnNormal;
}

int SystemModule::ToString_Eva(Object& ARGV)
{
  string content;
  for (auto i=1; i<= ARGV.Size(); i++ ){
    if ( ARGV[i].StringQ() ){
      content += ARGV[i].Key();
    }else{
      content += ARGV[i].ToString();
    }
  }
  ARGV.SetString( content );
  ReturnNormal;
}

/*
int SystemModule::ToExpression_Eva(Object& ARGV, Evaluation *Eva )
{
  CheckShouldEqual(1);
  CheckShouldBeString(1);
  Object t;
  t = ImportList::ToExpression( ARGV[1].Key() );
  ARGV = t;
  ReturnNormal;
}
*/

int SystemModule::MatchQ(Object & ARGV){
  CheckArgsShouldEqual(ARGV,2);
  EvaKernel->Evaluate(ARGV[1]);
  //cerr<<"ARGV = "<<ARGV.ToFullFormString()<<endl;
  //cerr<<"ARGV[1] = "<<ARGV[1].ToFullFormString()<<endl;
  Object&l=ARGV[1],&pattern = ARGV[2];
  Pattern::UnifyRuleLeft( pattern );
  ReturnBoolean( Pattern::UnifiedMatchQ( l, pattern ) );
}

int SystemModule::UnifiedMatchQ(Object & ARGV){
  CheckArgsShouldEqual(ARGV,2);
  Object&l=ARGV(1),&pattern = ARGV(2);
  ReturnBoolean( Pattern::UnifiedMatchQ(l,pattern));
}

int SystemModule::UnifyRule(Object & ARGV){
  CheckArgsShouldEqual(ARGV,1);
  Object&pattern = ARGV(1);
  if ( (pattern).Size() < 2 ){
    { _Erroring("UnifyRule","Rule form is right."); ReturnError; }
  }
  Pattern::UnifyRule( (pattern)[1], (pattern)[2] );
  ARGV = ARGV[1];
  ReturnNormal;
}

// int SystemModule::Hold(Object & ARGV){
//   for ( auto iter = ARGV.Begin(); iter!= ARGV.End(); iter++){
//     EvaKernel->Evaluate( *iter, true );
//   }
//   ReturnHold;
// }

// int SystemModule::HoldPattern(Object & ARGV){
//   CheckShouldEqual( 1 );
//   EvaKernel->Evaluate( ARGV[1], true );
//   ReturnHold;
// }

int SystemModule::HoldOnce(Object & ARGV){
  CheckShouldEqual(1);
  EvaKernel->Evaluate( ARGV[1], true );
  ARGV = ARGV[1];
  ReturnNormal;
}


int SystemModule::Replace(Object & ARGV){
  CheckArgsShouldEqual(ARGV,2);
  Object&l = ARGV(1);
  Object&rule = ARGV(2);
  Pattern::UnifyRule( rule );
  bool replaced = Pattern::UnifiedReplaceAll(l,rule,false);
  if ( replaced ) EvaKernel->Evaluate(l);
  ARGV = ARGV[1];
  ReturnNormal;
}

int SystemModule::ReplaceAll(Object & ARGV){
  CheckShouldEqual( 2);
  CheckShouldBeList( 2 );
  Object&l     = ARGV(1); 
  Object&rules = ARGV(2);
  bool replaced = true;
  if ( rules.PairQ( SYMID_OF_Rule ) or
       rules.PairQ( SYMID_OF_RuleDelayed ) or
       rules.PairQ( SYMID_OF_KeyValuePair ) ){
    Pattern::UnifyRule( rules );
    replaced = Pattern::UnifiedReplaceAll(l,rules,true);
    if ( replaced ) EvaKernel->Evaluate(l);
    ARGV = ARGV[1];
    ReturnNormal;
  }else{
    for (u_int i = 1; i<= rules.Size() ; i++){
      if ( not rules[i].PairQ() ) continue;
      Pattern::UnifyRule( rules[i] );
      if ( Pattern::UnifiedReplaceAll( l, rules[i] ,true) )
        replaced = true;
    }
    if ( replaced ) EvaKernel->Evaluate(l);
    ARGV = ARGV[1];
    ReturnNormal;
  }
  // _Erroring("ReplaceAll","Second parameter should be a Rule or a ARGV of rules.");
  // ReturnError; 
}

int SystemModule::ReplaceAllRepeated(Object & ARGV){
  CheckArgsShouldEqual(ARGV,2);
  Object&l = ARGV(1); 
  Object&rules = ARGV(2);
  bool replaced = true;
  if ( rules.PairQ( SYMID_OF_Rule ) or
       rules.PairQ( SYMID_OF_RuleDelayed ) or
       rules.PairQ( SYMID_OF_KeyValuePair ) ){
    Pattern::UnifyRule( rules );
    while ( replaced ){
      replaced = false;
      if ( Pattern::UnifiedReplaceAll(l, rules,true) ){
        replaced = true;
      }
      if ( replaced ) EvaKernel->Evaluate(l);
    }
  }else{
    for (u_int i = 1; i<= rules.Size() ; i++){
      if ( not rules[i].PairQ() ) continue;
      Pattern::UnifyRule( rules[i] );
    }
    while ( replaced ){
      replaced = false;
      for (u_int i = 1; i<= rules.Size() ; i++){
        if ( not rules[i].PairQ() ) continue;
        if ( Pattern::UnifiedReplaceAll( l, (rules)[i] ,true) )
          replaced = true;
      }
      if ( replaced ) EvaKernel->Evaluate(l);
    }
  }
  ARGV = ARGV[1];
  ReturnNormal;
}

int SystemModule::SimpleSimplify(Object&ARGV){
  if ( not ARGV.ListQ() )
    return 0;
  if ( ARGV.ListQ( SYMID_OF_SimpleSimplify ) ){
    CheckArgsShouldEqual(ARGV,1);
    ARGV = ARGV[1];
  }
  // return 0;
  //dout<<"try simplify "<<ARGV<<endl;
  static Object SSRules(ObjectType::List,SYMID_OF_List);
  // if ( false and SSRules.Size() == 0 ){
  if ( SSRules.Size() == 0 ){
#define addrule(rule_str) {                       \
      Object obj;                                 \
      obj =  ImportList::ToExpression(rule_str);  \
      Pattern::UnifyRule( obj );                  \
      SSRules.PushBackRef( obj );                 \
    }
    /////////////
    addrule("HoldPattern( Plus(a~))                                                 :> a");
    addrule("HoldPattern( Plus(a~~~,Null,b~~~))                                     :> Plus(a,b)");
    addrule("HoldPattern( Plus(a~~~,x~,x~,b~~~))                                    :> Plus(a,Times(2,x),b)");
    addrule("HoldPattern( Plus(a~~~,x~,Times(i~Number,x~),b~~~))                    :> Plus(a,Times(Plus(1,i),x),b)");
    addrule("HoldPattern( Plus(a~~~,Times(x~~),Times(i~Number,x~~),b~~~))           :> Plus(a,Times(Plus(1,i),x),b)");
    addrule("HoldPattern( Plus(a~~~,Times(i~Number,x~~),Times(j~Number,x~~),b~~~))  :> Plus(a,Times(Plus(i,j),x),b)");
    //// Times Power
    addrule("HoldPattern( Times(a~))                                                :> a");
    addrule("HoldPattern( Times(a~~~,,b~~~))                                        :> Times(a,b)");
    addrule("HoldPattern( Times(x~,x~))                                             :> Power(x,2)");
    addrule("HoldPattern( Times(a~~~,x~,x~,b~~~))                                   :> Times(a,Power(x,2),b)");
    addrule("HoldPattern( Times(a~~~,x~,Power(x~,i~Number),b~~~))                   :> Times(a,Power(x,Plus(1,i)),b)");
    addrule("HoldPattern( Times(a~~~,Power(x~,i~Number),Power(x~,j~Number),b~~~))   :> Times(a,Power(x,Plus(i,j)),b)");
    ////////////////////////
    //////////////////////// normal rules
    addrule("HoldPattern( Power(x~,0)                 :> 1 )");
    addrule("HoldPattern( Power(x~,1)                 :> x )");
    addrule("HoldPattern( Power(Power(x~,a~),b~)      :> Power(x,Times(a,b)) )");
    addrule("HoldPattern( Log(Power(x~,m~))           :> Times(m,Log(a)) )");
#undef addrule
  }
  //dout<<"simplify with rules:"<<SSRules<<endl;
  bool replaced = true;
  while (replaced){
    replaced = false;
    for ( u_int i= 1 ;i <= SSRules.Size() ; i++){
      if ( Pattern::UnifiedReplaceAll( ARGV , SSRules[i] ,true) )
        replaced = true;
    }
    if ( replaced ){
      EvaKernel->Evaluate( ARGV );
    }
  }
  return 0;
}

int SystemModule::GrammarModule(Object &ARGV){
  CheckArgsShouldNoLessThan(ARGV,2);
  // create the local variables;
  EvaKernel->newContext();
  Object variables = ARGV(1);
  if ( not (variables).ListQ( SYMID_OF_List ) ) {
    _Erroring("Module","Local variable should be in a List.");
    EvaKernel->deleteContext();
    ReturnError;
  }
  // dealing new variables in new context,
  // Part can be directly evaluated to the sybmol in the list
  for (Object::iterator iter = variables.Begin(); iter!=variables.End(); iter++){
    if ( (*iter).ListQ( SYMID_OF_Part ) ){
      Object&obj = *iter; 
      //cerr<<"use part as local var spec:"<<obj.ToFullFormString()<<endl;
      if ( obj.Size() < 2 ){
        _Erroring("Module","Local variable specification "+obj.ToString()+ " wrong.");
        EvaKernel->deleteContext();
        ReturnError;
      }
      Object result;
      //cerr<<"try get part result"<<endl;
      ListModule::GetPartList( obj[1], obj, 2, result );
      //cerr<<"result = "<< result << endl;
      if ( not result.SymbolQ() ){
        _Erroring("Module","Local variable specification "+result.ToString()+"("+obj.ToString()+ ") should be a symbol.");
        EvaKernel->deleteContext();
        ReturnError;
      }
      Object rvalue; rvalue.CopyObject(result);
      EvaKernel->InstantInsertOrUpdatePairValue(result,rvalue);
      continue;
    }
    if ( (*iter).SymbolQ() || (*iter).ListQ( SYMID_OF_Part ) ){
      Object rvalue; rvalue.CopyObject(*iter);
      //rvalue.KeyPrepend( "$" );
      EvaKernel->InstantInsertOrUpdatePairValue(*iter,rvalue);
      continue;
    }
    if ( (*iter).ListQ( SYMID_OF_Set ) ){
      EvaKernel->InstantInsertOrUpdatePairValue( (*iter)[1],(*iter)[2] );
      continue;
    }
    if ( (*iter).ListQ( SYMID_OF_SetDelayed ) ){
      //EvaKernel->Evaluate( (*iter)[2]); 
      EvaKernel->InstantInsertOrUpdatePairValue( (*iter)[1],(*iter)[2]);
      continue;
    }
    _Erroring("Module","Local variable specification "+variables.ToString()+" contains "+(*iter).ToString()+" which is not a atom symbol or an assignment to a symbol."); 
    EvaKernel->deleteContext();
    ReturnError;
  }
  //evaluate the returned ARGV
  for (Object::iterator iter = ARGV.Begin()+1; iter!=ARGV.End(); iter++){
    EvaKernel->Evaluate( *iter);
  }
  EvaKernel->deleteContext();
  ARGV = ARGV[ ARGV.Size() ];
  ReturnNormal;
}

int PureFunctionApply(Object&p_expr,Object&ARGV){
  if ( p_expr.NullQ() ) return 0;
  switch (p_expr.type() ){
  case ObjectType::Number: return 0;
  case ObjectType::String: return 0;
  case ObjectType::Symbol:{
    if ( p_expr.ids() == SYMID_OF_FunctionVariable ){
      u_int id = p_expr.re();
      if ( id == 0 or id > ARGV.Size() )
        { _Erroring("PureFunction","PureFunction variable id exceed the number of arguments."); ReturnError; }
      p_expr = ARGV[id];
    }
    return 0;
  }
  case ObjectType::List:{
    for (auto iter = p_expr.ElementsBegin(); iter != p_expr.ElementsEnd(); iter++){
      PureFunctionApply(*iter,ARGV);
    }
  }
  }
  return 0;
}

// pure function has no
int SystemModule::PureFunction(Object &ARGV){
  CheckShouldNoLessThan(1);
  CheckArgsShouldEqual(ARGV[0],1);
  Object expr = ARGV[0][1];
  PureFunctionApply(expr,ARGV);
  ARGV = expr;
  EvaKernel->Evaluate( ARGV );
  ReturnNormal;
}


int SystemModule::Function(Object &ARGV){
  // CheckShouldNoLessThan(1);
  Object&func = ARGV[0];
  CheckArgsShouldBeWithin(func,1,2);
  if ( func.Size() == 1 ){
    return PureFunction(ARGV);
  }
  Object &varlist=func[1];
  //dout<<"into function with varlist = "<<varlist <<endl;
  if ( varlist.SymbolQ() ){
    if ( ARGV.Size() != 1 ) {
      zhErroring("函数","参数列表和调用提供参数不一致.") ||
        _Erroring("Function","Function variable list is different from arguments applied to.");
      ReturnError;
    }
    EvaKernel->newContext();
    Object pairobj = EvaKernel->StackPushCopy( varlist, varlist);
    pairobj[2] = ARGV[1];
    Object res = func[2];
    EvaKernel->Evaluate( res );
    ARGV = res;
    EvaKernel->deleteContext();
    ReturnNormal;
  }else if ( varlist.ListQ() ){
    if ( ARGV.Size() != varlist.Size() ) {
      zhErroring("函数","参数列表和调用提供参数不一致.") ||
        _Erroring("Function","Function variable list is different from arguments applied to.");
      ReturnError;
    }
    EvaKernel->newContext();
    for(u_int i = 1; i <= varlist.Size() ; i++){
      if ( not varlist[i].SymbolQ() ) {
        zhErroring("函数","函数参数列表应该是一个符号列表") ||
          _Erroring("Function","Argument list elements are required to be Symbols.") ; 
        EvaKernel->deleteContext();
        ReturnError; 
      }
      Object pairobj = EvaKernel->StackPushCopy(varlist[i], varlist[i] );
      pairobj[2] = ARGV[i];
    }
    Object res = func[2];
    EvaKernel->Evaluate(res);
    ARGV = res;
    EvaKernel->deleteContext();
    ReturnNormal;
  }
  zhErroring("函数","函数参数列表应该是符号，或者符号列表") ||
    _Erroring("Function","Arguments should be symbols or list of symbols.") ; 
  ReturnError; 
}

int SystemModule::Conjunct(Object &ARGV){
  CheckShouldEqual(2);
  //Dealing 1st argument evaluate until to a conjunctable expr
  //cout<<"get into Conjunct with ARGV = "<<ARGV<<endl;
  Object &fobj = ARGV[1];
  //EvaKernel->Evaluate( fobj ); // should be reference evaluate
  EvaKernel->Evaluate( fobj, false, true );
  // and those obj with Conjunctable should take care of how to update the ref value
  //cout<<"fobj after eval= "<<fobj<<endl;
  if ( not fobj.SimpleListQ() ){
    ReturnHold;
  }
  if ( not fobj[0].ValuedSymbolQ() ){
    ReturnHold;
  }
  EvaRecord *rec = EvaKernel->GetEvaRecord( fobj[0] );
  if ( rec ){
    if ( AttributeQ(rec->attributes, AttributeType::Conjunctable ) ){
      // the conjunct pass through has form
      // Conjuct(  f(...), xxx )    xxx can be any form,  f is referenced value if is from value pair
      //dout<<"pass "<<ARGV << " to "<< fobj[0] <<endl;
      return EvaKernel->Call(rec, ARGV);
    }
    ReturnHold;
  }
  ReturnHold;
}

int SystemModule::If(Object &ARGV){
  CheckShouldBeWithin(2,3);
  EvaKernel->Evaluate(ARGV[1]);
  char res = ARGV[1].Boolean();
  if (  res > 0 ){
    EvaKernel->Evaluate(ARGV[2]);
    ARGV = ARGV[2];
  }else if (  res == 0 ){
    if ( ARGV.Size() == 3){
      EvaKernel->Evaluate(ARGV[3]);
      ARGV = ARGV[3];
    }else{// return Null
      ARGV.SetNull();
    }
  }
  // condition result not known Boolean() == -1
  //dprintf("If returned with hold");
  ReturnHold;
}
int SystemModule::While(Object &ARGV){
  CheckShouldEqual(2);
  Object&condition = ARGV(1);
  Object&expression = ARGV(2);
  Object cond,expr;
  for(;;){
    cond = NullObject;
    cond.CopyObject(condition);
    EvaKernel->Evaluate(cond);
    if ( cond.Boolean() > 0 ){ //evaluate expr
      expr = NullObject;
      expr.CopyObject(expression);
      EvaKernel->Evaluate( expr);
      continue;
    }
    break;
  }
  ReturnNull;
}

int SystemModule::For(Object &ARGV){
  CheckShouldEqual(4);
  Object&start= ARGV(1);
  EvaKernel->Evaluate( start);
  Object&test= ARGV(2);
  Object&incr= ARGV(3);
  Object&body= ARGV(4);
  Object ltest,lincr,lbody;
  for(;;){
    ltest = NullObject;
    ltest.CopyObject(test);
    EvaKernel->Evaluate( ltest);
    if ( ltest.Boolean() > 0 ){ //evaluate expr
      lbody = NullObject;
      lbody.CopyObject(body);
      EvaKernel->Evaluate( lbody);
      lincr = NullObject;
      lincr.CopyObject(incr);
      EvaKernel->Evaluate(lincr);
      continue;
    }
    break;
  }
  ReturnNull;
}

int SystemModule::Foreach(Object&ARGV){
  CheckShouldEqual(3);
  Object&vars  = ARGV[1];
  EvaKernel->Evaluate( ARGV[2] );
  Object&lists = ARGV[2];
  Object&expr  = ARGV[3];
  if ( not lists.ListQ() )
    { _Erroring("Foreach","Second argument is required to be a List."); ReturnError; }
  if ( vars.SymbolQ() ){
    int ptr = EvaKernel->GetStackPtr();
    Object pairobj = EvaKernel->StackPushCopy(vars,vars.Copy() );
    Object&vobj = pairobj[2];
    Object newexpr;
    // EvaKernel->PreEvaluate( expr );
    for (auto iter= lists.Begin(); iter != lists.End(); iter++){
      vobj.CopyObject( *iter );
      newexpr.CopyObject( expr );
      EvaKernel->Evaluate( newexpr );
    }
    EvaKernel->SetStackToPtr( ptr );
    ReturnNull;
  }else if ( vars.ListQ() ){
    int ptr = EvaKernel->GetStackPtr();
    Object vobj_list; vobj_list.SetList();
    for ( auto iter = vars.Begin(); iter != vars.End(); iter ++ ){
      Object pairobj = EvaKernel->StackPushCopy(*iter,*iter);
      vobj_list.PushBackRef( pairobj[2] );
    }
    Object newexpr;
    for ( auto iter = lists.Begin(); iter != lists.End(); iter++ ){
      if ( iter->Size() != vobj_list.Size() ) { 
        _Erroring("Foreach::shape","List shape is not consistent with variable list."); 
        EvaKernel->SetStackToPtr( ptr );
        ReturnError; 
      }
      for ( auto viter=vobj_list.Begin(),liter=(*iter).Begin();viter!=vobj_list.End();viter ++,liter++){
        (*viter).CopyObject( *liter );
      }
      newexpr = NullObject;
      newexpr.CopyObject( expr );
      EvaKernel->Evaluate( newexpr );
    }
    EvaKernel->SetStackToPtr( ptr );
    ReturnNull;
  }
  _Erroring("Foreach","Iterator should be a Symbol or a List of Symbols.");
  ReturnError; 
}

int SystemModule::Do(Object &ARGV){
  //dout<<"try deal with function Do"<<endl;
  CheckShouldNoLessThan(2);
  while ( ARGV.Size() >2 ){
    Object newtable; newtable.SetList( SYMID_OF_Do ); 
    newtable.PushBackRef( ARGV(1) );
    newtable.PushBackRef( ARGV(2) );
    ARGV.Delete(2);
    ARGV(1) = newtable;
  }
  Object&expr = ARGV(1);
  Object&iter = ARGV(2);
  if ( not (iter).ListQ() || iter.Size()<1 || iter.Size()>4) {
    _Erroring("Do", (iter).ToString()+" is not a valid iterator ARGV form."); 
    ReturnError; 
  }
  //iteratorToList(iter,newlist);
  if ( iter.Size() == 1 ){ // form: {100}
    Object&num = (iter)(1);
    EvaKernel->Evaluate(num);
    if ( num.NumberQ() ){
      int N = num.Number() ;
      Object newexpr; 
      // EvaKernel->PreEvaluate( expr );
      for ( int i = 1; i<=N; i++ ){
        newexpr = NullObject;
        newexpr.CopyObject(expr );
        EvaKernel->Evaluate(newexpr);
      }
      ARGV.SetNull();
      ReturnNormal ;
    }
    _Erroring(ARGV.Key(),"Iterator "+iter.ToString()+" does not have appropriate bounds."); 
    ReturnError; 
  }
  if ( iter.Size() == 2 ){ // form: {i, 100}
    Object&var = (iter)(1);
    if ( !var.SymbolQ() ) { _Erroring("Do",var.ToString()+" cannot be used as an iterator."); ReturnError; }
    Object&num = (iter)(2);
    EvaKernel->Evaluate(num);
    //dout<<"try push stack var:"<< var<<" to table:"<<(EvaKernel->currentValueTable->stackTable)<<endl;
    Object&vobj = EvaKernel->StackPushCopy(var,var)[2];
    if ( num.NumberQ() ){
      int N =  num.Number();
      Object newexpr;
      // EvaKernel->PreEvaluate( expr );
      for (auto i = 1; i<=N; i++ ){
        newexpr = NullObject;
        newexpr.CopyObject( expr );
        vobj.SetNumber(i);
        EvaKernel->Evaluate(newexpr);
      }
      EvaKernel->StackPop();
      ReturnNull;
    }else if ( (num).ListQ() ){
      Object newexpr;
      // EvaKernel->PreEvaluate( expr );
      for (auto i = 1; i<=num.Size(); i++ ){
        newexpr = NullObject;
        newexpr.CopyObject( expr );
        vobj =  num[i];
        EvaKernel->Evaluate(newexpr);
      }
      EvaKernel->StackPop();
      ReturnNull;
    }
    _Erroring(ARGV.Key(),"Iterator "+iter.ToString()+" does not have appropriate bounds."); 
    ReturnError;
  }
  if ( iter.Size() == 3 ){ // for {i, 1 , 100}
    Object&var = (iter)(1);
    if ( !var.SymbolQ() ) { _Erroring("Do",var.ToString()+" cannot be used as an iterator."); ReturnError; }
    Object&num1 = (iter)(2);
    Object&num2 = (iter)(3);
    EvaKernel->Evaluate(num1);
    EvaKernel->Evaluate(num2);
    if ( num1.NumberQ() && num2.NumberQ() ){
      int Nbegin =  num1.Number();
      int Nend =  num2.Number();
      Object newexpr;
      Object pairobj = EvaKernel->StackPushCopy(var,var);
      Object&vobj = pairobj[2];
      // EvaKernel->PreEvaluate( expr );
      for ( int i = Nbegin; i<=Nend; i++ ){
        newexpr = NullObject;
        newexpr.CopyObject(expr);
        vobj.SetNumber(i);
        EvaKernel->Evaluate(newexpr);
      }
      EvaKernel->StackPop();
      ReturnNull;
    }
    { _Erroring(ARGV.Key(),"Iterator "+iter.ToString()+" does not have appropriate bounds."); ReturnError; }
  }
  if ( iter.Size() == 4 ){ // form :  {i,1,100,3}
    Object var = (iter)(1);
    if ( !var.SymbolQ() ) { 
      _Erroring(ARGV.Key(),var.ToString()+" cannot be used as an iterator."); 
      ReturnError; 
    }
    Object num1 = (iter)(2);
    Object num2 = (iter)(3);
    Object incr = (iter)(4);
    EvaKernel->Evaluate(num1);
    EvaKernel->Evaluate(num2);
    EvaKernel->Evaluate(incr);
    if ( num1.NumberQ() && num2.NumberQ() && incr.NumberQ() ){
      double Nbegin = num1.Number( );
      double Nend = num2.Number( );
      double Nincr = incr.Number( );
      if ( Nincr == 0 ) { _Erroring("Do","Increment should not be a zero."); ReturnError; }
      int steps = (Nend-Nbegin)/Nincr;
      Object newexpr;
      Object pairobj = EvaKernel->StackPushCopy(var,var);
      Object&vobj = pairobj[2];
      // EvaKernel->PreEvaluate( expr );
      for ( int i = 0 ;i<=steps;i++){
        newexpr = NullObject;
        newexpr.CopyObject(expr);
        vobj.SetNumber(Nbegin+i*Nincr);
        EvaKernel->Evaluate(newexpr);
      }
      EvaKernel->StackPop();
      ReturnNull;
    }
    _Erroring(ARGV.Key(),"Iterator "+iter.ToString()+" does not have appropriate bounds."); 
    ReturnError;
  }
  ReturnHold;
}

int SystemModule::ToExpression(Object & ARGV){
  string s;
  if ( not ARGV.StringQ() ){
    CheckArgsShouldEqual(ARGV,1);
    EvaKernel->Evaluate(ARGV[1]);
    CheckArgsShouldBeString(ARGV,1);
    s = ARGV[1].Key();
  }else{
    s = ARGV.Key();
  }
  Object tobj = ImportList::ToExpression(s);
  if ( tobj.voidQ() )
    { _Erroring("ToExpression","Ivalid string to transform to POND expression."); ReturnError; }
  ARGV = tobj;
  EvaKernel->Evaluate(ARGV);
  ReturnNormal;
}

int system_with_print(string cmd, vector<string>&result){
  result.clear();
  FILE *fp;
  char line[1024];
  fp = popen(cmd.c_str(),"r");
  if ( fp == NULL ){
    _Erroring("System",cmd+" cannot be excuted");
    return -1;
  }
  while ( fgets(line,sizeof(line),fp) != NULL ){
    if ( line[ strlen(line)-1 ] == '\n' ){
      line[ strlen(line)-1 ] = '\0';
    }
    result.push_back(line);
  }
  pclose(fp);
  return result.size();
}

int SystemModule::PD_System(Object &ARGV){
  CheckArgsShouldEqual(ARGV,1);
  EvaKernel->Evaluate( ARGV[1] );
  if ( !ARGV[1].StringQ() ) { _Erroring(ARGV.Key(),"First argument should be a string."); ReturnError; }
  vector<string> strs;
  if ( system_with_print( ARGV[1].Key(),strs ) <0 )
    { _Erroring(ARGV.Key(),"cmd "+ARGV[1].ToString()+" execute error."); ReturnError; }
  ARGV.SetList();
  for ( int i=0; i<(int)strs.size(); i++){
    ARGV.PushBackString( strs[i].c_str() );
  }
  ReturnNormal;
}

int SystemModule::SystemWithoutOutput(Object &ARGV)
{
  CheckArgsShouldEqual(ARGV,1);
  EvaKernel->Evaluate( ARGV[1] );
  if ( !ARGV[1].StringQ() ) { _Erroring(ARGV.Key(),"First argument should be a string."); ReturnError; }
  if ( system( ARGV[1].Key() ) <0 )
    { _Erroring(ARGV.Key(),"cmd "+ARGV[1].ToString()+" execute error."); ReturnError; }
  ReturnNull;
}

// const u_char Set_Temp_State = 1;
int SystemModule::Set( Object&ARGV){
  //if not symbol then evaluate its head such as a[x],,, a == Dict or Matrix or List or something
  CheckShouldEqual(2);
  Object&left = ARGV[1];
  Object&right = ARGV[2];
  // if ( ARGV.state() != Set_Temp_State ){
  EvaKernel->Evaluate( right );
  // }
  // not only left need to be evaluated, but also symbols
  //dout<<"get into set with ARGV = "<<ARGV<<endl;
  if ( left.ListQ() ){ // evaluate  in reference style
    EvaKernel->Evaluate(left[0],false,true);
  }
  if ( left.ListQ(SYMID_OF_List) ){
    if ( right.ListQ(SYMID_OF_List) ){
      if ( left.Size() != right.Size() ) {
        zhErroring("赋值","列表大小不匹配.") ||
          _Erroring("Set","Left List size should match size of right List.");
        ReturnError;
      }
      for ( u_int i = 1; i <= left.Size(); i++ ){
        Object tSet; tSet.SetList( SYMID_OF_Set );
        tSet.set_state( 1 );
        tSet.PushBackRef( left[i] ); tSet.PushBackRef( right[i] );
        Set( tSet );
      }
    }else{
      for ( u_int i = 1; i <= left.Size(); i++ ){
        Object tSet; tSet.SetList( SYMID_OF_Set );
        tSet.PushBackRef( left[i] ); tSet.PushBackRef( right );
        Set( tSet );
      }
    }
    // if ( ARGV.state() != Set_Temp_State ){
    //   ARGV.SetVoid();
    //   // ARGV = ARGV[2];
    // }
    ReturnObject( ARGV[2] );
  }
  //normal
  if ( left.NumberQ() or left.StringQ()  ){
    zhErroring("赋值","不能对数字或者字符串赋值.") ||
      _Erroring("Set","Left value of set is not allowed to be a Number or String.");
    ReturnError; 
  }
  //check if the left is a Setable or DelaySetable List
  if ( left.ListQ() ){
    //Conjunct
    // if left is a conjuntable object then it has already get its valuetable ref for
    // previous evaluation in reference style 
    if ( left[0].SymbolQ( SYMID_OF_Conjunct ) ){ // Conjunct
      //  case of:   A.a = b
      //dout<<"unprocessed conjunct ARGV = "<<ARGV<<endl;
      Object&conLeft = left[1];
      Object&conRight = left[2];
      EvaKernel->Evaluate( conLeft, false, true );
      //dout<<"unprocessed conjunct ARGV after eval conLeft= "<<ARGV<<endl;
      if ( conLeft.SimpleListQ() and conLeft[0].ValuedSymbolQ() ){
        EvaRecord *rec = EvaKernel->GetEvaRecord( conLeft[0] );
        if ( rec ){
          if ( AttributeQ( rec->attributes , AttributeType::Setable ) ){
            return rec->Call( ARGV );
          }else{
            zhErroring("赋值",conLeft.ToString()+" 不可赋值，因为 "+conLeft[0].ToString()+" 不具备可赋值属性") ||
              _Erroring("Set",conLeft.ToString()+" can not be assigned because "+conLeft[0].ToString()+" does not have Setable Attribute.");
            ReturnHold;
          }
        }
      }
      // left.CopyObject( right );
      // if ( ARGV.state() != Set_Temp_State ) 
      //   ARGV = ARGV[2];
      ReturnNormal;
    }else{
      // Setable
      // EvaKernel->Evaluate( left, false, true ); // ref evaluate without hold
      // so everyfunction that 
      if ( left[0].SymbolQ() ) { //and left[0].idx() != Index0 ){ // a vlaued symbol
        EvaRecord * rec = EvaKernel->GetEvaRecord( left[0] );
        if ( rec ){
          if ( AttributeQ( rec->attributes , AttributeType::Setable ) ){
            // the conjunct pass through has form
            // Set(  f(...), xxx )    xxx can be any form,  f is referenced value if is from value pair
            //dout<<" pass "<<ARGV<< " to "<<left[0]<<endl;
            return rec->Call( ARGV );
          }
          // else{
          //   zhErroring("赋值",left.ToString()+" 不可赋值，因为 "+left[0].ToString()+" 不具备可赋值属性") ||
          //     _Erroring("Set",left.ToString()+" can not be assigned because "+left[0].ToString()+" does not have Setable Attribute.");
          //   ReturnHold;
          // }
        }
      }
      //DelaySetable
      if ( left[0].SimpleListQ() ){ // and left[0][0].idx() != Index0 ){
        EvaRecord * rec = EvaKernel->GetEvaRecord( left[0][0] );
        if ( rec ){
          if ( AttributeQ(rec->attributes ,AttributeType::DelaySetable) ){
            // the conjunct pass through has form
            // Set(  f(...)(...), xxx )    xxx can be any form,  f is referenced value if is from value pair
            //dout<<" pass "<<ARGV<< " to "<<left[0]<<endl;
            return rec->Call( ARGV );
          }
          // }else{
          //   zhErroring("赋值",left[0][0].ToString()+" 不具备可延迟赋值属性") ||
          //     _Erroring("Set",left[0][0].ToString()+" does not have DelaySetable Attribute.");
          //   ReturnHold;
          // }
        }
      }
    }
  }
  //get left
  //check whether pattern
  // if left is  symbol it has not been evaluate yet, previous ref eval is only for list
  Object res = EvaKernel->GetOrNewValuePairRef(left); // the List pair
  if ( not res.NullQ() ){//is not a pattern
    if ( res.ids() != 0 ){//check whether protected
      if (  EvaKernel->AttributeQ(res.ids() ,AttributeType::Protected) ){
        zhErroring("赋值",(left).ToString() +" 处于被保护状态.")||
          _Erroring("Set",(left).ToString() +" is Protected.");
        ReturnError;
      }
    }
    res[2].CopyObject(right);
  }else{//is a pattern
    EvaKernel->RefInsertOrUpdatePairValue(left,right);
  }
  // if ( ARGV.state() != Set_Temp_State )
  //   ARGV = ARGV[2];
  ReturnObject( ARGV[2] );
  // ARGV.SetVoid();
  // return 1;
}

int SystemModule::SetDelayed(Object & ARGV){
  CheckShouldEqual(2);
  Object&left = ARGV(1);  
  Object&right = ARGV(2);
  if ( left.ValuedSymbolQ() ){
    Object( left.idx() )[2] =  right;
    ReturnNull;
  }
  CheckShouldNotBeNumber(1);
  CheckShouldNotBeString(1);
  //get left
  //check whether pattern
  if ( not Pattern::PatternListQ(left ) ){
    Object res = EvaKernel->GetOrNewValuePairRef(left,false,true);
    if ( res.ids() != 0 ){//check whether protected
      if (  EvaKernel->AttributeQ( res.ids(),AttributeType::Protected) ){
        zhErroring("Set",(left).ToString() +(string)" 处于被保护状态, 不能被赋值.") ||
          _Erroring("Set",(left).ToString() +(string)" is Protected."); 
        ReturnError; 
      }
    }
    res[2] = right;
    ReturnNull;
  }else{//is a pattern
    EvaKernel->InsertOrUpdatePairValue(left,right,false,true);
    ReturnNull;
  }
}

int SystemModule::ShowModuleFunctions(Object &ARGV){
  CheckArgsShouldEqual(ARGV,1);
  CheckShouldBeString(1);
  ReturnNormal;
}

int SystemModule::ShowModules(Object &ARGV){
  CheckArgsShouldEqual(ARGV,0);
  ReturnNormal;
}

int SystemModule::SetOutputFormat(Object &ARGV){
  CheckShouldBeWithin(1,2);
  CheckShouldBeNumber(1);
  if ( ARGV.Size() == 2){
    CheckShouldBeNumber(2);
    pond::SetPrintFormat((double)ARGV[1],(double)ARGV[2]);
  } else {
    pond::SetPrintFormat((double)ARGV[1]);
  }
  ReturnNull;
}

int SystemModule::SystemArgv(Object &ARGV)
{
  CheckShouldNoMoreThan(1);
  if ( ARGV.Size() == 0 ){
    //EvaKernel->argc;
    ARGV.SetList();
    for (int i=0; i<EvaKernel->argc; i++ ){
      ARGV.PushBackString( EvaKernel->argv[i] );
    }
    ReturnNormal;
  }
  CheckShouldBeNumber(1);
  int p = (int)ARGV[1];
  if ( p < 0 ) { _Erroring("SystemArgv","Input should be a non-negative Number."); ReturnError; }
  if ( p >= EvaKernel->argc ) ReturnString("");
  ReturnString( EvaKernel->argv[p] );
}

int SystemModule::ShowHelpInfo(Object &ARGV){
  CheckShouldEqual(1);
  EvaRecord*rec = EvaKernel->GetEvaRecord(ARGV(1));
  if ( rec == NULL ){
    cout<<"No help info found for "<<ARGV[1].ToString()<<endl;
  }else{
    cout<<rec->description<<endl;
  }
  ReturnNull;
}

int SystemModule::Clear(Object&ARGV){
  if ( ARGV.Size() == 0 ) ReturnNull;
  for ( u_int i=1;i<=ARGV.Size();i++){
    if ( not ARGV[i].SymbolQ() ) {
      zhErroring("Clear","Clear的参数应该是一个符号.") ||
        _Erroring("Clear","Arguments of Clear is required to be Symbol."); 
      ReturnError; 
    }
    if ( EvaKernel->Clear(ARGV[i]) < 0 ) {
      zhErroring("清除",ARGV[i].ToString()+" 处于被保护中.")||
        _Erroring("Clear",(string)ARGV[i].Key()+" is Protected.");
      ReturnError;
    }
  }
  ReturnNull;
}

int SystemModule::AbsoluteTime(Object &ARGV){
  struct timeval tvs;
  gettimeofday(&tvs,NULL);
  ARGV.SetNumber( tvs.tv_sec+tvs.tv_usec/1000000.0);
  ReturnNormal;
}

int SystemModule::CpuTiming(Object &ARGV){
  CheckShouldEqual(1);
  clock_t start,finish;
  start = clock();
  EvaKernel->Evaluate(ARGV[1]);
  finish = clock();
  ARGV[0].SetSymbol( SYMID_OF_List );
  ARGV.InsertNumber( 1,( (1.0*finish-start)/CLOCKS_PER_SEC ) );
  ReturnNormal;
}

int SystemModule::Timing(Object &ARGV){
  CheckShouldEqual(1);
  struct timeval start,finish;
  gettimeofday( &start,NULL );
  EvaKernel->Evaluate(ARGV[1]);
  gettimeofday( &finish,NULL );
  ARGV[0].SetSymbol( SYMID_OF_List );
  ARGV.InsertNumber( 1,finish.tv_sec - start.tv_sec + (finish.tv_usec - start.tv_usec)/1000000.0 );
  ReturnNormal;
}

int SystemModule::DateList(Object &ARGV){
  ARGV.SetList();
  ReturnNormal;
}

int SystemModule::Sleep(Object&ARGV){
  // if ( ARGV.NumberQ() ){
  //   pond::sleep_ms( ARGV.Number()*1000 );
  // }
  CheckShouldEqual(1);
  CheckShouldBeNumber(1);
  pond::sleep_micro( ARGV[1].Number()*1000 );
  ReturnNull;
}


int func_else(Object&ARGV,bool condi){ // if ( a , else(b,c) )
  //dout<<"into func_else "<<ARGV<<", condi =" <<condi<<endl;
  if ( ARGV.Size() != 2 ) return -1;
  if ( condi > 0 ){ // if ( a , else(b,c) )  a is true
    EvaKernel->Evaluate( ARGV[1] );
    ARGV = ARGV[1];
    ReturnNormal;
  }else{ // if (a, else(b,c) )   a is false
    EvaKernel->Evaluate( ARGV[2] );
    ARGV = ARGV[2];
    ReturnNormal;
  }
}

int func_elif(Object&ARGV,bool condi){ // if (a, elif(b,(c),d) )
  //dout<<"into func_elif "<<ARGV<<", condi =" <<condi<<endl;
  if ( ARGV.Size() != 3 ) return -1;
  if ( condi > 0 ){ // if (a , elif(b,c,d) )   a is true, return evaluated b
    EvaKernel->Evaluate( ARGV[1] );
    ARGV = ARGV[ 1 ];
    ReturnNormal;
  }else{ // need to check if else or elif
    if ( not ARGV[2].ListQ( SYMID_OF_Parenthesis ) ){
      zhErroring("elif","elif 的条件项应该放在 () 中" ) ||
        _Erroring("elif","elif condition should enclosed with ()" );
      ReturnError;
    }
    EvaKernel->Evaluate( ARGV[2] ); // evaluate c
    char res = ARGV[2].Boolean();
    //dout<<" cal own condi to "<<int(res)<<endl;
    if ( res < 0 ){ // cannot evaluate, rewrite to if(c,d)
      ARGV.Delete(1);
      ARGV[0].SetSymbol( SYMID_OF_if );
      return res;
    }

    if ( ARGV[3].ListQ( SYMID_OF_else ) ){ // d is else
      res = func_else( ARGV[3], res );
      ARGV = ARGV[3];
      return res;
    }else if ( ARGV[3].ListQ( SYMID_OF_elif ) ){ // d is elif
      res = func_elif( ARGV[3], res );
      ARGV = ARGV[3];
      return res;
    }else{ //  d is normal expression
      if ( res > 0 ){
        EvaKernel->Evaluate(ARGV[3]);
        ARGV = ARGV[3];
        ReturnNormal;
      }
      ReturnNull;
    }
  }
}

int SystemModule::PD_if(Object&ARGV){
  /*zh:
    ----------------------
    if ( condition ){
      expr_true;
    }
    ----------------------
    if ( condition ){
      expr_true;
    } else {
      expr_false;
    }
    ----------------------
   */
  CheckShouldEqual(2);
  if ( not ARGV[1].ListQ( SYMID_OF_Parenthesis ) ){
    zhErroring("if","if 条件项应该放在 () 中" ) ||
      _Erroring("if","if condition should enclosed with ()" );
    ReturnError;
  }
  EvaKernel->Evaluate(ARGV[1]);
  char res = ARGV[1].Boolean();
  if ( res < 0  ) ReturnHold; 
  if ( ARGV[2].ListQ( SYMID_OF_else ) ){ // if (a, else(b,c) )
    res = func_else( ARGV[2], res );
    ARGV = ARGV[2];
    return res;
  }else if ( ARGV[2].ListQ( SYMID_OF_elif ) ){ // if(a elsif(b,c,d) )
    res = func_elif( ARGV[2], res );
    ARGV = ARGV[2];
    return res;
  }else{ // if(a,b),  b is simple expression
    if ( res > 0 ){
      EvaKernel->Evaluate( ARGV[2] );
      ARGV = ARGV[2];
      ReturnNormal;
    }else if ( res == 0 ){
      ReturnNull;
    }
  }
  ReturnHold;
}

int SystemModule::PD_while(Object&ARGV) {
  CheckShouldEqual(2);
  if ( not ARGV[1].ListQ( SYMID_OF_Parenthesis ) ){
    zhErroring("while","while 后面应该紧跟括号.") ||
      _Erroring("while","while should be followed with ().") ;
  }
  Object&condition = ARGV(1);
  Object&body = ARGV(2);
  Object cond, expr, bodyelse;
  if ( body.ListQ(SYMID_OF_else) and body.Size() == 2 ){
    bodyelse = body[2];
    body     = body[1];
  }
  for(;;){
    cond.CopyObject( condition );
    EvaKernel->Evaluate( cond );
    if ( cond.Boolean() > 0 ){ //evaluate expr
      expr.CopyObject( body );
      EvaKernel->Evaluate( expr );
      // return
      if ( EvaKernel->statusCode == 88 ){ARGV.SetVoid(); ReturnNormal;}
      if ( EvaKernel->statusCode == 66 ){EvaKernel->statusCode = 0; continue;}
      if ( EvaKernel->statusCode == 44 ){
        bodyelse = NullObject;
        EvaKernel->statusCode = 0;
        break;
      }
      continue;
    }
    break;
  }
  if ( not bodyelse.NullQ() ) EvaKernel->Evaluate( bodyelse );
  ReturnNull;
}

static INIT_SYMID_OF(range);
int SystemModule::PD_for(Object&ARGV){
  //dout<< "dealing for "<< ARGV <<endl;
  CheckShouldEqual(2);
  if ( not ARGV[1].ListQ( SYMID_OF_Parenthesis ) ){
    zhErroring("for","for 后面应该紧跟括号.") ||
      _Erroring("for","for should be followed with ().");
  }
  if ( ARGV[1].Size() < 1 ){
    zhErroring("for","for 括号中参数太少") ||
      _Erroring("for","for has too few arguments in().");
  }
  //dout<<"for ARGV[1]"<<ARGV<<endl;
  if ( ( ARGV[1][1].ListQ( SYMID_OF_in ) )
         and ARGV[1][1].Size() == 2 ){ // for ( i in {1,2,3,4} ){ expr }
    Object&in = ARGV(1)(1);
    Object&var = in[1];
    Object&body= ARGV(2);
    Object bodyelse;
    if ( body.ListQ( SYMID_OF_else ) and body.Size() == 2 ){
      bodyelse = body[2];
      body = body[1];
    }

    if ( in[2].ListQ(SYMID_OF_range) ){
      // as single variable interate
      if ( not var.SymbolQ() ){
        zhErroring("for:in:range","循环变量应该是一个符号.") ||
          _Erroring("for:in:range","the iterator variable should be a symbol.");
      }
      double start, incr; long N;
      if ( ListModule::range_iter(in[2],N,start,incr) == 0 ){
        zhErroring("for:in:range","range 不合法") ||
          _Erroring("for:in:range","range is not valid.");
        ReturnError;
      }
      Object&vobj = EvaKernel->StackPushCopy(var,var)[2];
      Object newexpr;
      // EvaKernel->PreEvaluate( body );
      for(long i=0; i<N; i++ ){
        vobj.SetNumber( start+i*incr );
        newexpr.CopyObject( body , true);
        EvaKernel->Evaluate( newexpr );
        if ( EvaKernel->statusCode == 88 ){ARGV.SetVoid(); ReturnNormal;}
        if ( EvaKernel->statusCode == 66 ){EvaKernel->statusCode = 0; continue;}
        if ( EvaKernel->statusCode == 44 ){
          bodyelse = NullObject;
          EvaKernel->statusCode = 0;
          break;
        }
      }
      if ( not bodyelse.NullQ() ) EvaKernel->Evaluate( bodyelse );
      EvaKernel->StackPop();
      ReturnNull;
    }else{
      //dout<<"deal for in with in = "<<in<<endl;
      EvaKernel->Evaluate( in[2] );
      //dout<<"after deal in = "<<in<<endl;
      Object&lists = in[2];
      if ( not lists.ListQ() ){
        zhErroring("for","in 后面的参数应该是一个列表.") ||
          _Erroring("for","argument after in is required to be a List.");
        ReturnError;
      }

      if ( var.SymbolQ() ){
        Object&vobj = EvaKernel->StackPushCopy(var,var)[2];
        Object newexpr;
        for (auto iter= lists.Begin(); iter != lists.End(); iter++){
          vobj =  *iter ;
          newexpr.CopyObject( body );
          EvaKernel->Evaluate( newexpr );
          // return continue break from pond
          if ( EvaKernel->statusCode == 88 ){ARGV.SetVoid(); ReturnNormal;}
          if ( EvaKernel->statusCode == 66 ){EvaKernel->statusCode = 0; continue;}
          if ( EvaKernel->statusCode == 44 ){
            bodyelse = NullObject;
            EvaKernel->statusCode = 0;
            break;
          }
        }
        if ( not bodyelse.NullQ() ) EvaKernel->Evaluate( bodyelse );
        EvaKernel->StackPop();
        ReturnNull;
      }
      if ( var.ListQ() ){
        int ptr = EvaKernel->GetStackPtr();
        Object vobj_list; vobj_list.SetList();
        for ( auto iter = var.Begin(); iter != var.End(); iter ++ ){
          vobj_list.PushBackRef(  EvaKernel->StackPushCopy(*iter,*iter)[2] );
        }
        Object newexpr;
        for ( auto iter = lists.Begin(); iter != lists.End(); iter++ ){
          if ( iter->Size() != vobj_list.Size() ) { 
            zhErroring("for::in::shape","列表和变量列表的形状不一致.") ||
              _Erroring("for::in::shape","List shape is not consistent with variable list."); 
            ReturnError; 
          }
          for ( auto viter=vobj_list.Begin(),liter=(*iter).Begin();viter!=vobj_list.End();viter ++,liter++){
            (*viter).CopyObject( *liter );
          }
          newexpr = NullObject;
          newexpr.CopyObject( body );
          EvaKernel->Evaluate( newexpr );
          // return continue break from pond
          if ( EvaKernel->statusCode == 88 ){ARGV.SetVoid(); ReturnNormal;}
          if ( EvaKernel->statusCode == 66 ){EvaKernel->statusCode = 0; continue;}
          if ( EvaKernel->statusCode == 44 ){
            bodyelse = NullObject;
            EvaKernel->statusCode = 0;
            break;
          }
        }
        if ( not bodyelse.NullQ() ) EvaKernel->Evaluate( bodyelse );
        EvaKernel->SetStackToPtr( ptr );
      }
    }
    zhErroring("for:in","variable should be a symbol or list of symbols.")||
      _Erroring("for:in","变量应该是一个符号变量或者一个由符号变量构成的列表."); 
    ReturnHold;
  }
  zhErroring("for","for clause should be in form for ( i in [n1,n2,...nN] ) expr;")||
    _Erroring("for","for 语句应该形为 for ( i in [n1,n2,...nN] ) expr;"); 
  ReturnHold;
}

int SystemModule::PD_in(Object&ARGV){
  //"test if one obj is in list" ){ // a in {...} a in [...]    or  a in (...)
  CheckShouldEqual(2);
  //dout<<"deal in with ARGV = "<< ARGV<<endl;
  bool exist = false;
  Object &list = ARGV[2];
  EvaKernel->Evaluate( list, false, true );
  CheckShouldBeList(2);
  Object &var  = ARGV[1];
  EvaKernel->Evaluate( var, false, true );
  if ( list.ListQ( SYMID_OF_Dict ) ){
    Object res = list.DictGetPair( var );
    if ( res.NullQ() ) ReturnNumber(SYMID_OF_False);
    ReturnNumber(SYMID_OF_True);
  }
  for (auto iter = list.Begin(); iter != list.End(); iter ++ ){
    if  ( var == (*iter) ){
      ReturnSymbol(SYMID_OF_True);
    }
  }
  ReturnSymbol(SYMID_OF_False );
}

int SystemModule::PD_RunPageJS(Object&ARGV){
  //"在前端页面直接执行JS代码"
  // in form RunJS[ " console.log("this is good"); ]
  
  ReturnNormal;
}

int SystemModule::PD_RunKernelJS(Object&ARGV){ //"在运算单元中直接执行JS代码"
  // in form RunJS[ " console.log("this is good"); ]
  
  ReturnNormal;
}

int SystemModule::PD_continue(Object&ARGV){//"结束本轮循环，并开始下一轮循环" 
  EvaKernel->statusCode = 66;
  ReturnNull;
}

int SystemModule::PD_break(Object&ARGV){ //, "跳出当前循环"
  EvaKernel->statusCode = 44;
  ReturnNull;
}

int SystemModule::PD_return(Object&ARGV){ //"将当前值作为当前代码域返回值并返回" 
  /*
    Attributes: HoldAll
   */
  //dout<<"get into return ARGV = "<<ARGV<<endl;
  EvaKernel->statusCode = 88;
  if ( ARGV.Size() >= 1 ){
    ARGV = ARGV[ ARGV.Size() ];
  }else{
    ARGV.SetVoid();
  }
  EvaKernel->statusObject = ARGV;
  ReturnNormal;
}

int function_def_process(Object&ARGV){ //, "通用函数定义" 
  // used in form:   function f(a,b,x,y=1,z=3){...}       inner form ...(fname, (a,b...), {...} )
  // return or trandi way to return value
  // *args, **kwargs
  CheckShouldEqual(2);
  if ( not ARGV[1].ListQ() or not ARGV[1][0].SymbolQ() ){
    zhErroring(ARGV[0].ToString(),"函数声明格式错误: "+ARGV[1].ToString() )||
      _Erroring(ARGV[0].ToString(),"function declaration is in the wrong form: "+ARGV[1].ToString() );
    ReturnError;
  }
  CheckShouldBeListWithHead(2, SYMID_OF_ExpressionList );
  //dout<<"into def with ARGV ="<<ARGV<<endl;
  // deal those with default values
  Object & args = ARGV[1];
  Object dict(__List__,SYMID_OF_Dict);
  Object packVar;
  Object dictPackVar;
  int i = 1;
  while ( i<= args.Size() ){
    if ( args[i].SymbolQ() ){
      if ( not packVar.NullQ() ){
        zhErroring(ARGV[0].ToString(),"位置参数出现在参数包后") ||
          _Erroring(ARGV[0].ToString(),"Positional argument follows arguments pack");
        ReturnError;
      }
      if ( not dictPackVar.NullQ() ){
        zhErroring(ARGV[0].ToString(),"位置参数出现在字典参数包后") ||
          _Erroring(ARGV[0].ToString(),"Positional argument follows keyword arguments pack");
        ReturnError;
      }
    }else if ( args[i].PairQ( SYMID_OF_Set ) ){
      EvaKernel->Evaluate( args[i][2] );
      // dict.DictInsertPairRef( args[i] );
      Object::iterator iter;
      bool res = dict.DictGetPosition( args[i][1], iter ); 
      if ( res == 0 ){
        zhErroring(ARGV[0].ToString(),"字典参数 "+args[i][1].ToString()+" 被重复指定") ||
          _Erroring(ARGV[0].ToString(),"Multiple Keyword argument "+args[i][1].ToString()+" specified.");
        ReturnError;
      }
      dict.InsertRef(iter, args[i] );
      i++; //args.Delete(i);
      continue;
    }else if ( args[i].PairQ( SYMID_OF_SetDelayed ) ){
      zhErroring(ARGV[0].ToString(),"延迟赋值不能用来给参数提供默认值") ||
        _Erroring(ARGV[0].ToString(),"SetDelayed can not be used to provide default value");
      ReturnError;
    }else if ( args[i].ListQ( SYMID_OF_Unpack )  ){
      if ( not packVar.NullQ() ){
        zhErroring(ARGV[0].ToString(),"参数包只能又一个") ||
          _Erroring(ARGV[0].ToString(),"Multi argument pack");
        ReturnError;
      }
      if ( args[i].Size() != 1 and not args[i][1].SymbolQ() ){
        zhErroring(ARGV[0].ToString(),"参数包应该具有符号名字") ||
          _Erroring(ARGV[0].ToString(),"Arguments pack should has a symbol name");
        ReturnError;
      }
      packVar = args[i][1];
      //dout<<"try delete pack var at "<<i <<" args ="<<args<<endl;
      args.Delete(i);
      continue;
    }else if ( args[i].ListQ( SYMID_OF_UnpackDict )  ){
      if ( not dictPackVar.NullQ() ){
        zhErroring(ARGV[0].ToString(),"字典参数包只能有一个") ||
          _Erroring(ARGV[0].ToString(),"Multi keyword argument pack");
        ReturnError;
      }
      if ( args[i].Size() != 1 and not args[i][1].SymbolQ() ){
        zhErroring(ARGV[0].ToString(),"字典参数包应该具有符号名字") ||
          _Erroring(ARGV[0].ToString(),"Dict arguments pack should has a symbol name");
        ReturnError;
      }
      dictPackVar = args[i][1];
      //dout<<"try delete dict pack var at "<<i <<" args ="<<args<<endl;
      args.Delete(i);
      continue;
    }else{
      zhErroring(ARGV[0].ToString(),args[i].ToString() + "不能用来作为参数") ||
        _Erroring(ARGV[0].ToString(),args[i].ToString() + " can not be used to as argument");
      ReturnError;
    }
    i++;
  }
  ARGV.PushBackRef(dict);
  ARGV.PushBackRef(packVar);
  ARGV.PushBackRef(dictPackVar);
  // save processed value to valueTable
  ARGV[0].SetSymbol( SYMID_OF_FUNCTION$DEFINED$ );
  return 1;
}

int SystemModule::PD_def(Object&ARGV){ //, "通用函数定义" 
  function_def_process( ARGV );
  // // used in form:   function f(a,b,x,y=1,z=3){...}       inner form ...(fname, (a,b...), {...} )
  // // return or trandi way to return value
  // // *args, **kwargs
  // CheckShouldEqual(2);
  // if ( not ARGV[1].ListQ() or not ARGV[1][0].SymbolQ() ){
  //   zhErroring(ARGV[0].ToString(),"函数声明格式错误: "+ARGV[1].ToString() )||
  //     _Erroring(ARGV[0].ToString(),"function declaration is in the wrong form: "+ARGV[1].ToString() );
  //   ReturnError;
  // }
  // CheckShouldBeListWithHead(2, SYMID_OF_ExpressionList );
  // //dout<<"into def with ARGV ="<<ARGV<<endl;
  // // deal those with default values
  // Object & args = ARGV[1];
  // Object dict(__List__,SYMID_OF_Dict);
  // Object packVar;
  // Object dictPackVar;
  // int i = 1;
  // while ( i<= args.Size() ){
  //   if ( args[i].SymbolQ() ){
  //     if ( not packVar.NullQ() ){
  //       zhErroring(ARGV[0].ToString(),"位置参数出现在参数包后") ||
  //         _Erroring(ARGV[0].ToString(),"Positional argument follows arguments pack");
  //       ReturnError;
  //     }
  //     if ( not dictPackVar.NullQ() ){
  //       zhErroring(ARGV[0].ToString(),"位置参数出现在字典参数包后") ||
  //         _Erroring(ARGV[0].ToString(),"Positional argument follows keyword arguments pack");
  //       ReturnError;
  //     }
  //   }else if ( args[i].PairQ( SYMID_OF_Set ) ){
  //     EvaKernel->Evaluate( args[i][2] );
  //     // dict.DictInsertPairRef( args[i] );
  //     Object::iterator iter;
  //     bool res = dict.DictGetPosition( args[i][1], iter ); 
  //     if ( res == 0 ){
  //       zhErroring(ARGV[0].ToString(),"字典参数 "+args[i][1].ToString()+" 被重复指定") ||
  //         _Erroring(ARGV[0].ToString(),"Multiple Keyword argument "+args[i][1].ToString()+" specified.");
  //       ReturnError;
  //     }
  //     dict.InsertRef(iter, args[i] );
  //     i++; //args.Delete(i);
  //     continue;
  //   }else if ( args[i].PairQ( SYMID_OF_SetDelayed ) ){
  //     zhErroring(ARGV[0].ToString(),"延迟赋值不能用来给参数提供默认值") ||
  //       _Erroring(ARGV[0].ToString(),"SetDelayed can not be used to provide default value");
  //     ReturnError;
  //   }else if ( args[i].ListQ( SYMID_OF_Unpack )  ){
  //     if ( not packVar.NullQ() ){
  //       zhErroring(ARGV[0].ToString(),"参数包只能又一个") ||
  //         _Erroring(ARGV[0].ToString(),"Multi argument pack");
  //       ReturnError;
  //     }
  //     if ( args[i].Size() != 1 and not args[i][1].SymbolQ() ){
  //       zhErroring(ARGV[0].ToString(),"参数包应该具有符号名字") ||
  //         _Erroring(ARGV[0].ToString(),"Arguments pack should has a symbol name");
  //       ReturnError;
  //     }
  //     packVar = args[i][1];
  //     //dout<<"try delete pack var at "<<i <<" args ="<<args<<endl;
  //     args.Delete(i);
  //     continue;
  //   }else if ( args[i].ListQ( SYMID_OF_UnpackDict )  ){
  //     if ( not dictPackVar.NullQ() ){
  //       zhErroring(ARGV[0].ToString(),"字典参数包只能有一个") ||
  //         _Erroring(ARGV[0].ToString(),"Multi keyword argument pack");
  //       ReturnError;
  //     }
  //     if ( args[i].Size() != 1 and not args[i][1].SymbolQ() ){
  //       zhErroring(ARGV[0].ToString(),"字典参数包应该具有符号名字") ||
  //         _Erroring(ARGV[0].ToString(),"Dict arguments pack should has a symbol name");
  //       ReturnError;
  //     }
  //     dictPackVar = args[i][1];
  //     //dout<<"try delete dict pack var at "<<i <<" args ="<<args<<endl;
  //     args.Delete(i);
  //     continue;
  //   }else{
  //     zhErroring(ARGV[0].ToString(),args[i].ToString() + "不能用来作为参数") ||
  //       _Erroring(ARGV[0].ToString(),args[i].ToString() + " can not be used to as argument");
  //     ReturnError;
  //   }
  //   i++;
  // }
  // ARGV.PushBackRef(dict);
  // ARGV.PushBackRef(packVar);
  // ARGV.PushBackRef(dictPackVar);
  // // save processed value to valueTable
  // ARGV[0].SetSymbol( SYMID_OF_FUNCTION$DEFINED$ );
  EvaKernel->InstantInsertOrUpdatePairValue(ARGV[1][0], ARGV);
  ReturnNull;
}

#define _err_return ({ ARGV[0] = ARGV[0][1][0]; ReturnError;})
int SystemModule::PD_FUNCTION$DEFINED$(Object&ARGV){
  // 函数调用 DelayFunction
  //dout<<"FUNCTION$DEFINED$ called with ARGV="<<ARGV<<endl;
  //EvaKernel->GetMemInfo();
  // check function form
  // argv should be evaluated before newContext


  Object pairTable(__List__,SYMID_OF_Dict);
  Object &func = ARGV[0];
  if ( not func.ListQ() or func.Size() != 5 or
       not func[3].ListQ() or  // dict, {} will read into expression list
       not ( func[4].NullQ() or func[4].SymbolQ() ) or
       not ( func[5].NullQ() or func[5].SymbolQ()  )
  ){
    zhErroring("函数调用","非法的函数调用")||
      _Erroring("DefinedFunction","Invalid function call.");
    _err_return;
  }

  //dout<<" try create new Context with func="<<func<<endl;
  Object &funcargs = func[1];
  Object &funcbody = func[2];
  Object &dict = func[3];
  //dout<<"current dict is "<<dict<<endl;
  //dout<<"current pairTable "<<pairTable<<endl;
  Object &packVar     = func[4];
  Object &dictPackVar = func[5];
  // match and create vars 
  //check if positional argument follows keyword argument
  bool isDict = false;
  Object pack;
  if ( not packVar.NullQ() ){
    Object pair(__List__,SYMID_OF_KeyValuePair);
    pack.SetList( SYMID_OF_List);
    pair.SetPairRef( packVar, pack ); 
    pairTable.DictInsertOrUpdatePairRef( pair );
  }
  Object packDict;
  if ( not func[5].NullQ() ){
    Object pair(__List__,SYMID_OF_KeyValuePair);
    packDict.SetList( SYMID_OF_Dict );
    pair.SetPairRef( dictPackVar, packDict ); 
    pairTable.DictInsertOrUpdatePairRef( pair );
  }

  //dout<<"argv process prepare done"<<endl;
  //EvaKernel->GetMemInfo();
  //dout<<"try deal real argument one by one"<<endl;
  //int pos = 1;
  for(int i = 1; i <= ARGV.Size(); i++ ){
    //dout<<"deal argv[ "<<i<<"] = "<<ARGV[i]<<endl;
    if ( ARGV[i].ListQ( SYMID_OF_Unpack ) ){
      //dout<<"deal unpack"<<ARGV[i]<<endl;
      EvaKernel->Evaluate( ARGV[i] );
      ARGV.InsertRef(ARGV.Begin()+i, ARGV[i].Begin(), ARGV[i].End() );
      ARGV.Delete(i);
      if ( i>ARGV.Size() ) break;
    }else if ( ARGV[i].ListQ( SYMID_OF_UnpackDict ) ){ // deal one by one key maybe string
      //dout<<"deal unpackdict "<<ARGV[i]<<endl;
      EvaKernel->Evaluate( ARGV[i] );
      if ( not ARGV[i].ListQ( SYMID_OF_Sequence ) ){
        zhErroring("函数调用",funcargs[0].ToString()+"() 试图对非字典数据进行字典解包")||
          _Erroring("Function call",funcargs[0].ToString()+"() try to dict-unpack none dict type.");
        _err_return;
      }
      ARGV.InsertRef(ARGV.Begin()+i, ARGV[i].Begin(), ARGV[i].End() );
      ARGV.Delete(i);
      if ( i>ARGV.Size() ) break;
    }// unpack will insert new arguments inpalce which will processed by following logic
    
    // keyword argument
    if ( ARGV[i].PairQ( SYMID_OF_Set ) or ARGV[i].PairQ( SYMID_OF_KeyValuePair ) ){
      //dout<<"deal keyword arg "<<i<<" = "<<ARGV[i]<<endl;
      isDict = true;
      Object::iterator iter;
      //dout<<"try insert to dict"<<endl;
      if ( ARGV[i][1].StringQ() ) ARGV[i][1].SetSymbol( ARGV[i][1].Key() );

      bool res = pairTable.DictGetPosition( ARGV[i][1], iter ); 
      //dout<<"find result ="<<res<<endl;
      if ( res == 0 ){ // find position
        zhErroring("函数调用",funcargs[0].ToString()+"() 重复获得字典参数 "+ARGV[i][1].ToString()+"的值")||
          _Erroring("Function call",funcargs[0].ToString()+"() got multiple values for keyword argument "+ARGV[i][1].ToString()+"");
        _err_return;
      }else{
        // find if a keyword para
        Object::iterator iterdic;
        res = dict.DictGetPosition( ARGV[i][1], iterdic );
        if ( res == 0 ){
          //dout<<"is a keyword para"<<endl;
          EvaKernel->Evaluate( ARGV[i][2] );
          pairTable.InsertRef(iter, ARGV[i] );
          dict.Delete( iterdic );
          continue;
        }
        // find if a possition para
        //dout<<"check if a positional argument"<<endl;
        for (int j = i; j <= funcargs.Size(); j++ ){
          if ( funcargs[j].SymbolQ() and ARGV[i][1] == funcargs[j] ){
            EvaKernel->Evaluate( ARGV[i][2] );
            pairTable.InsertRef(iter, ARGV[i] );
            funcargs.Delete( j );
            continue;
          }
        }
        // unexpected maybe if no packDict exists
      }
      if ( packDict.NullQ() ){
        zhErroring("函数调用",funcargs[0].ToString()+"() 遇到预期之外的参数 "+ARGV[i].ToString() )||
          _Erroring("DefinedFunction",funcargs[0].ToString()+"() called with unexpected arugment: "+ARGV[i].ToString() );
        _err_return;
      }
      EvaKernel->Evaluate( ARGV[i][2] );
      Object pair(__List__,SYMID_OF_KeyValuePair);
      pair.SetPairRef( Object(__String__,ARGV[i][1].Key() ), ARGV[i][2] );
      packDict.DictInsertPairRef( pair );
      continue;

    }else{ // positional argument
      //dout<<"deal positional argument "<<i<<" = "<<ARGV[i]<<endl;
      if ( isDict ){
        zhErroring("函数调用",funcargs[0].ToString()+"() 位置参数处于字典参数之后")||
          _Erroring("DefinedFunction",funcargs[0].ToString()+"() called with positional argument follows  kwyword argument");
        _err_return;
      }
      if ( i <= funcargs.Size() ){
        EvaKernel->Evaluate( ARGV[i] );
        if ( funcargs[i].SymbolQ() ){// one one positional corresponding
          //dout<<"try insert to dict"<<endl;
          Object pair;   pair.SetPairRef( funcargs[i] , ARGV[i] );
          pairTable.DictInsertOrUpdatePairRef( pair );
          //dout<<"insert done"<<endl;
          continue;
        }else{ // corresponding position is a  positional argument, while it cor. to a optional argument
          //dout<<" positional input "<<ARGV[i]<<" to optional arg "<<funcargs[i]<<endl;
          Object::iterator iter;
          bool res = pairTable.DictGetPosition( funcargs[i][1], iter ); 
          if ( res == 0 ){
            zhErroring("函数调用",funcargs[0].ToString()+"() 重复获得字典参数 "+ARGV[i][1].ToString()+"的值")||
              _Erroring("Function call",funcargs[0].ToString()+"() got multiple values for keyword argument "+ARGV[i][1].ToString()+"");
            _err_return;
          }
          Object pair;   pair.SetPairRef( funcargs[i][1], ARGV[i] );
          //dout<<"try insertt "<<pair<<" to pairTable "<<pairTable<<endl;
          pairTable.InsertRef( iter, pair );
          //dout<<"after insert pairTable ="<<pairTable<<endl;

          Object::iterator iterdic;
          dict.DictGetPosition( funcargs[i][1], iterdic );
          dict.Delete( iterdic );
          continue;
        }
      }
      // extra real positional arguments, push to packVar list, or report error
      if ( pack.NullQ() ){
        zhErroring("函数调用",funcargs[0].ToString()+"() 调用使用了额外的位置参数")||
          _Erroring("DefinedFunction",funcargs[0].ToString()+"() called with extra positional argument");
        _err_return;
      }
      EvaKernel->Evaluate( ARGV[i] );
      pack.PushBackRef( ARGV[i] );
      continue;
    }

  }
  //dout<<"arguments are processed"<<endl;
  //dout<< "args done with isDict="<<isDict<<" funcargs="<<funcargs<<" ARGV="<<ARGV<<endl;
  //dout<<"dict="<<dict<<endl;
  //dout<<"pairTable="<<pairTable<<endl;
  //EvaKernel->GetMemInfo();

  //dout<<"process argv left"<<endl;

  if ( ARGV.Size() < funcargs.Size() and not funcargs[ ARGV.Size()+1 ].PairQ() ){
    zhErroring("函数调用",funcargs[0].ToString()+"() 未提供值给 "+funcargs[ARGV.Size()+1].ToString() )||
      _Erroring("DefinedFunction",funcargs[0].ToString()+"() called without value for "+funcargs[ARGV.Size()+1].ToString() );
    _err_return;
  }
  //dout<<"try insert extra optional argument values into pairTable dict="<<dict<<endl;
  // insert default values to pairTable
  while ( dict.Size() > 0 ){
    pairTable.DictInsertPairRef( dict.Last() );
    dict.PopBack();
  }
  //dout<<"args in ARGV done with new ARGV to be "<< ARGV<<endl;

  //dout<<"dict="<<dict<<endl;
  //dout<<"pairTable="<<pairTable<<endl;
  //dout<<"funcbody="<<funcbody<<endl;
  //dout<<"arguments are processed try new context"<<endl;
  // EvaKernel->GetMemInfo();

  // ReturnHold;
  EvaKernel->newContext(pairTable);
  //dout<<"funcbody = "<<funcbody<<endl;
  // EvaKernel->GetMemInfo();
  EvaKernel->Evaluate( funcbody );
  //dout<<"funcbody = "<<funcbody<<endl;
  // EvaKernel->GetMemInfo();
  if ( EvaKernel->statusCode == 88 ){
    //dout<<"value is return by return clause"<<endl;
    ARGV = EvaKernel->statusObject;
  }else{ // no return just return the default result
    //dout<<"value is return by normal evaluation"<<endl;
    ARGV =  funcbody;
    //dout<<"value by func body "<<ARGV<<" objid ="<<ARGV.objid<<endl;
  }
  EvaKernel->deleteContext();
  //dout<<"after delete Context()"<<endl;
  // EvaKernel->GetMemInfo();
  EvaKernel->statusCode = 0;
  EvaKernel->statusObject = NullObject;
  //dout<<"value is return by normal evaluation "<<ARGV<<" objid ="<<ARGV.objid<<","<<EvaKernel->statusObject.objid<<endl;
  ReturnNormal;
}
#undef _err_return

int SystemModule::PD_Unpack(Object&ARGV){ //, "Unpack list to its parent list"
  CheckShouldEqual(1);
  CheckShouldBeList(1);
  //dout<<"try to unpack "<< ARGV<<endl;
  ARGV[1][0].SetSymbol( SYMID_OF_Sequence ); 
  ARGV = ARGV[1];
  ReturnNormal;
}

int SystemModule::PD_UnpackDict(Object&ARGV){ //, "Unpack list to its parent list"
  CheckShouldEqual(1);
  CheckShouldBeList(1);
  //dout<<"try to unpack "<< ARGV<<endl;
  ARGV[1][0].SetSymbol( SYMID_OF_Sequence ); 
  ARGV = ARGV[1];
  ReturnNormal;
}


int SystemModule::PD_var(Object&args){
  /*
    将变量声明为本地变量
    var x = 99,y=100
  */
  for (int i = 1; i<= args.Size(); i++ ){
    Object pair(__List__,SYMID_OF_Set);
    if ( args[i].SymbolQ() ){
      pair.SetPairRef( args[i], args[i].Copy() );
      EvaKernel->StackPushPairRef( pair );
    }else if ( args[i].ListQ( SYMID_OF_Part ) ){
      Object result;
      ListModule::GetPartList( args[i][1], args[i], 2, result );
      if ( not result.SymbolQ() ){
        zhErroring("var","本地变量定义 "+result.ToString()+" 需要是一个符号.") ||
          _Erroring("var","Local variable specification "+result.ToString()+ " should be a symbol.");
        ReturnError;
      }
      pair.SetPairRef(result, result.Copy() );
      EvaKernel->StackPushPairRef( pair );
    }else if ( args[i].PairQ(SYMID_OF_Set ) ){
      if ( args[i][1].SymbolQ() ){
        EvaKernel->Evaluate( args[i][2] );
        EvaKernel->StackPushPairRef( args[i] );
      }else if ( args[i][1].ListQ(SYMID_OF_Part) ){ //
        Object result;
        ListModule::GetPartList( args[i][1][1], args[i][1], 2, result );
        if ( not result.SymbolQ() ){
          zhErroring("var","本地变量定义 "+result.ToString()+" 需要是一个符号.") ||
            _Erroring("var","Local variable specification "+result.ToString()+ " should be a symbol.");
          ReturnError;
        }
        pair.SetPairRef( result, args[i][2] );
        EvaKernel->StackPushPairRef( pair );
      }
    }else{
      zhErroring("本地变量声明","错误的局部变量定义表达式"+args[i].ToString() ) ||
        _Erroring("var","invalid local variable defination expression "+args[i].ToString() );
      ReturnError;
    }
  }
  ReturnNull;
}

int SystemModule::PD_context(Object&args){
  /*
    将其中代码置于新的上下文中运行
   */
  if ( args.Size() == 1 and args[1].ListQ( SYMID_OF_ExpressionList ) ){
    args = args[1];
  }else{
    args[0].SetSymbol( SYMID_OF_ExpressionList );
  }
  EvaKernel->newContext();
  int res = EvaKernel->Evaluate( args );
  if ( EvaKernel->statusCode == 88 ){
    args = EvaKernel->statusObject;
  }
  EvaKernel->statusCode = 0;
  EvaKernel->statusObject = NullObject;
  EvaKernel->deleteContext();
  return res;
}

int SystemModule::PD_type(Object&args){
  CheckShouldEqual(1);
  switch( args[1].type() ){
  case ObjectType::Number : ReturnSymbol( "Number" );
  case ObjectType::String : ReturnSymbol( "String" );
  case ObjectType::Symbol : ReturnSymbol( "Symbol" );
  case ObjectType::List   : ReturnSymbol( "List"   );
  }
  zhErroring("Object::type","") ||
    _Erroring("Object::type","");
  ReturnError;
}

int SystemModule::PD_ArrowFunction(Object&ARGV){
  //dout<<"into arrow function "<<ARGV<<endl;
  ARGV[1][0].SetSymbol(SYMID_OF_ArrowFunction);
  function_def_process(ARGV);
  //dout<<"processed to"<<ARGV<<endl;
  return 1;
}

int SystemModule::PD_SetParallelMode(Object&Argv)
{
  CheckShouldEqual(1);
  CheckShouldBeString(1);
  if ( strcasecmp( Argv[1].Key(), "gpu" ) == 0 ){
    pond::SetParallelMode( ParallelModeGpu );
  }else if ( strcasecmp( Argv[1].Key(), "cpu" ) == 0 ){
    pond::SetParallelMode( ParallelModeCpu );
  }else{
    ThrowError("SetParallelMode","Running mode "+Argv[1].ToString()+" is not recongnized.");
  }
  ReturnNull;
}

int SystemModule::PD_GetParallelMode(Object&Argv)
{
  CheckShouldEqual(0);
  if ( pond::GetParallelMode() == ParallelModeGpu ){
    ReturnString("gpu");
  }
  ReturnString("cpu");
}

int SystemModule::PD_GPUDeviceQ(Object&Argv)
{
  if ( pond::Execute("nvidia-smi -L >/dev/null 2>&1") != 0 ){
    ReturnString("No nVIDIA GPU Driver found.");
  }
  vector<string> res;
  pond::System("nvidia-smi -L",res) ;
  Argv.SetList();
  for ( auto li: res){
    Argv.PushBackString( li);
  }
  ReturnNormal;
}

int SystemModule::PD_SetCudaThreadsNumberPerBlock(Object&Argv)
{
  CheckShouldEqual(1);
  CheckShouldBeNumber(1);
  int num = (int)Argv[1];
  if ( num <= 0 and num%32 != 0 and num > 1024 )
    ThrowError("SetCudaThreadsNumberPerBlock","Thread number per block is suggested to be multiple of 32 and a number with 0 and 1024.");
  pond::SetThreadNumberPerBlock( num );
  ReturnNull;
}

int SystemModule::PD_SetCpuThreadsNumber(Object&Argv)
{
  CheckShouldEqual(1);
  CheckShouldBeNumber(1);
  int num = (int)Argv[1];
  if ( num <= 0 )
    ThrowError("SetEvawizCpuKernelThreadsNumber","Thread number should be a positive number.");
  pond::SetThreadNumberPerKernel( num );
  ReturnNull;
}

int SystemModule::PD_GetCudaThreadsNumberPerBlock(Object&Argv)
{
  CheckShouldEqual(0);
  ReturnNumber( EvaSettings.threadNumberPerBlock );
}

int SystemModule::PD_GetCpuThreadsNumber(Object&Argv)
{
  CheckShouldEqual(0);
  ReturnNumber( EvaSettings.threadNumberPerKernel );
}
